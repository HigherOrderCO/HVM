#include "hip/hip_runtime.h"
#define INTERPRETED
#define WITH_MAIN

#include "alloc.cuh"
#include "structs/tm.cuh"
#include "structs/book.cuh"
#include "structs/gnet.cuh"
#include "structs/rbag.cuh"
#include "numb.cuh"
#include "show.cuh"
#include "sync.cuh"

#include <stdio.h>
#include <stdlib.h>

// Linking
// -------

// Atomically Links `A ~ B`.
__device__ void link(Net* net, TM* tm, Port A, Port B) {
  #ifdef DEBUG
  Port INI_A = A;
  Port INI_B = B;
  #endif

  u32 lps = 0;

  // Attempts to directionally point `A ~> B`
  while (true) {

    // If `A` is NODE: swap `A` and `B`, and continue
    if (get_tag(A) != VAR && get_tag(B) == VAR) {
      Port X = A; A = B; B = X;
    }

    // If `A` is NODE: create the `A ~ B` redex
    if (get_tag(A) != VAR) {
      //printf("[%04x] new redex A %s ~ %s\n", GID(), show_port(A).x, show_port(B).x);
      push_redex(tm, new_pair(A, B)); // TODO: move global ports to local
      break;
    }

    // While `B` is VAR: extend it (as an optimization)
    B = enter(net, B);

    // Since `A` is VAR: point `A ~> B`.
    if (true) {
      // If B would leak...
      if (is_global(A) && is_local(B)) {
        // If B is a var, just swap it
        if (is_var(B)) {
          Port X = A; A = B; B = X;
          continue;
        }
        // If B is a nod, create a leak interaction
        if (is_nod(B)) {
          //if (!TID()) printf("[%04x] NODE LEAK %s ~ %s\n", GID(), show_port(A).x, show_port(B).x);
          push_redex(tm, new_pair(A, B));
          break;
        }
      }

      // Sanity check: if global A is unfilled, delay this link
      if (is_global(A) && vars_load(net, get_val(A)) == 0) {
        push_redex(tm, new_pair(A, B));
        break;
      }

      // Stores `A -> B`, taking the current `A` subst as `A'`
      Port A_ = vars_exchange(net, get_val(A), B);

      // If there was no `A'`, stop, as we lost B's ownership
      if (A_ == NONE) {
        break;
      }

      #ifdef DEBUG
      if (A_ == 0) printf("[%04x] ERR LINK %s ~ %s | %s ~ %s\n", GID(), show_port(INI_A).x, show_port(INI_B).x, show_port(A).x, show_port(B).x);
      #endif

      // Otherwise, delete `A` (we own both) and link `A' ~ B`
      vars_take(net, get_val(A));
      A = A_;
    }
  }
}

// Links `A ~ B` (as a pair).
__device__ void link_pair(Net* net, TM* tm, Pair AB) {
  link(net, tm, get_fst(AB), get_snd(AB));
}

// Interactions
// ------------

// The Link Interaction.
__device__ bool interact_link(Net* net, TM* tm, Port a, Port b) {
  // If A is a global var and B is a local node, leak it:
  // ^A ~ (b1 b2)
  // ------------- LEAK-NODE
  // ^X ~ b1
  // ^Y ~ b2
  // ^A ~ ^(^X ^Y)
  if (is_global(a) && is_nod(b) && is_local(b)) {
    // Allocates needed nodes and vars.
    if (!get_resources(net, tm, 3, 0, 0)) {
      return false;
    }

    tm->leak += 1;

    // Loads ports.
    Pair l_b  = node_take(net, get_val(b));
    Port l_b1 = enter(net, get_fst(l_b));
    Port l_b2 = enter(net, get_snd(l_b));

    // Leaks port 1.
    Port g_b1;
    if (is_local(l_b1)) {
      g_b1 = new_port(VAR, g_vars_alloc_1(net));
      vars_create(net, get_val(g_b1), NONE);
      link_pair(net, tm, new_pair(g_b1, l_b1));
    } else {
      g_b1 = l_b1;
    }

    // Leaks port 2.
    Port g_b2;
    if (is_local(l_b2)) {
      g_b2 = new_port(VAR, g_vars_alloc_1(net));
      vars_create(net, get_val(g_b2), NONE);
      link_pair(net, tm, new_pair(g_b2, l_b2));
    } else {
      g_b2 = l_b2;
    }

    // Leaks node.
    Port g_b = new_port(get_tag(b), g_node_alloc_1(net));
    node_create(net, get_val(g_b), new_pair(g_b1, g_b2));
    link_pair(net, tm, new_pair(a, g_b));

    return true;

  // Otherwise, just perform a normal link.
  } else {
    // Allocates needed nodes and vars.
    if (!get_resources(net, tm, 1, 0, 0)) {
      return false;
    }

    link_pair(net, tm, new_pair(a, b));
  }

  return true;
}

// Declared here for use in call interactions.
static inline bool interact_eras(Net* net, TM* tm, Port a, Port b);

// The Call Interaction.
#ifdef COMPILED
///COMPILED_INTERACT_CALL///
#else
__device__ bool interact_eras(Net* net, TM* tm, Port a, Port b);
__device__ bool interact_call(Net* net, TM* tm, Port a, Port b) {
  // Loads Definition.
  u32 fid  = get_val(a) & 0xFFFFFFF;
  Def* def = &BOOK.defs_buf[fid];

  // Copy Optimization.
  if (def->safe && get_tag(b) == DUP) {
    return interact_eras(net, tm, a, b);
  }

  // Allocates needed nodes and vars.
  if (!get_resources(net, tm, def->rbag_len + 1, def->node_len, def->vars_len)) {
    return false;
  }

  // Stores new vars.
  for (u32 i = 0; i < def->vars_len; ++i) {
    vars_create(net, tm->vloc[i], NONE);
  }

  // Stores new nodes.
  for (u32 i = 0; i < def->node_len; ++i) {
    node_create(net, tm->nloc[i], adjust_pair(net, tm, def->node_buf[i]));
  }

  // Links.
  for (u32 i = 0; i < def->rbag_len; ++i) {
    link_pair(net, tm, adjust_pair(net, tm, def->rbag_buf[i]));
  }
  link_pair(net, tm, new_pair(adjust_port(net, tm, def->root), b));

  return true;
}
#endif

// The Void Interaction.
__device__ bool interact_void(Net* net, TM* tm, Port a, Port b) {
  return true;
}

// The Eras Interaction.
__device__ bool interact_eras(Net* net, TM* tm, Port a, Port b) {
  // Allocates needed nodes and vars.
  if (!get_resources(net, tm, 2, 0, 0)) {
    return false;
  }

  // Loads ports.
  Pair B  = node_take(net, get_val(b));
  Port B1 = get_fst(B);
  Port B2 = get_snd(B);

  // Links.
  link_pair(net, tm, new_pair(a, B1));
  link_pair(net, tm, new_pair(a, B2));

  return true;
}

// The Anni Interaction.
__device__ bool interact_anni(Net* net, TM* tm, Port a, Port b) {
  // Allocates needed nodes and vars.
  if (!get_resources(net, tm, 2, 0, 0)) {
    return false;
  }

  // Loads ports.
  Pair A  = node_take(net, get_val(a));
  Port A1 = get_fst(A);
  Port A2 = get_snd(A);
  Pair B  = node_take(net, get_val(b));
  Port B1 = get_fst(B);
  Port B2 = get_snd(B);

  // Links.
  link_pair(net, tm, new_pair(A1, B1));
  link_pair(net, tm, new_pair(A2, B2));

  return true;
}

// The Comm Interaction.
__device__ bool interact_comm(Net* net, TM* tm, Port a, Port b) {
  // Allocates needed nodes and vars.
  if (!get_resources(net, tm, 4, 4, 4)) {
    return false;
  }

  // Loads ports.
  Pair A  = node_take(net, get_val(a));
  Port A1 = get_fst(A);
  Port A2 = get_snd(A);
  Pair B  = node_take(net, get_val(b));
  Port B1 = get_fst(B);
  Port B2 = get_snd(B);

  // Stores new vars.
  vars_create(net, tm->vloc[0], NONE);
  vars_create(net, tm->vloc[1], NONE);
  vars_create(net, tm->vloc[2], NONE);
  vars_create(net, tm->vloc[3], NONE);

  // Stores new nodes.
  node_create(net, tm->nloc[0], new_pair(new_port(VAR, tm->vloc[0]), new_port(VAR, tm->vloc[1])));
  node_create(net, tm->nloc[1], new_pair(new_port(VAR, tm->vloc[2]), new_port(VAR, tm->vloc[3])));
  node_create(net, tm->nloc[2], new_pair(new_port(VAR, tm->vloc[0]), new_port(VAR, tm->vloc[2])));
  node_create(net, tm->nloc[3], new_pair(new_port(VAR, tm->vloc[1]), new_port(VAR, tm->vloc[3])));

  // Links.
  link_pair(net, tm, new_pair(new_port(get_tag(b), tm->nloc[0]), A1));
  link_pair(net, tm, new_pair(new_port(get_tag(b), tm->nloc[1]), A2));
  link_pair(net, tm, new_pair(new_port(get_tag(a), tm->nloc[2]), B1));
  link_pair(net, tm, new_pair(new_port(get_tag(a), tm->nloc[3]), B2));

  return true;
}

// The Oper Interaction.
__device__ bool interact_oper(Net* net, TM* tm, Port a, Port b) {
  // Allocates needed nodes and vars.
  if (!get_resources(net, tm, 1, 1, 0)) {
    return false;
  }

  // Loads ports.
  Val  av = get_val(a);
  Pair B  = node_take(net, get_val(b));
  Port B1 = get_fst(B);
  Port B2 = enter(net, get_snd(B));

  // Performs operation.
  if (get_tag(B1) == NUM) {
    Val  bv = get_val(B1);
    Numb cv = operate(av, bv);
    link_pair(net, tm, new_pair(new_port(NUM, cv), B2));
  } else {
    node_create(net, tm->nloc[0], new_pair(a, B2));
    link_pair(net, tm, new_pair(B1, new_port(OPR, tm->nloc[0])));
  }

  return true;
}

// The Swit Interaction.
__device__ bool interact_swit(Net* net, TM* tm, Port a, Port b) {
  // Allocates needed nodes and vars.
  if (!get_resources(net, tm, 1, 2, 0)) {
    return false;
  }

  // Loads ports.
  u32  av = get_u24(get_val(a));
  Pair B  = node_take(net, get_val(b));
  Port B1 = get_fst(B);
  Port B2 = get_snd(B);

  // Stores new nodes.
  if (av == 0) {
    node_create(net, tm->nloc[0], new_pair(B2, new_port(ERA,0)));
    link_pair(net, tm, new_pair(new_port(CON, tm->nloc[0]), B1));
  } else {
    node_create(net, tm->nloc[0], new_pair(new_port(ERA,0), new_port(CON, tm->nloc[1])));
    node_create(net, tm->nloc[1], new_pair(new_port(NUM, new_u24(av-1)), B2));
    link_pair(net, tm, new_pair(new_port(CON, tm->nloc[0]), B1));
  }

  return true;
}

// Pops a local redex and performs a single interaction.
__device__ bool interact(Net* net, TM* tm, Pair redex, u32 turn) {
  // Gets redex ports A and B.
  Port a = get_fst(redex);
  Port b = get_snd(redex);

  // Gets the rule type.
  Rule rule = get_rule(a, b);

  // If there is no redex, stop.
  if (redex != 0) {
    //if (GID() == 0 && turn == 0x201) {
      //Pair kn = get_tag(b) == CON ? node_load(net, get_val(b)) : 0;
      //printf("%04x:[%04x] REDUCE %s ~ %s | par? %d | (%s %s)\n",
        //turn, GID(),
        //show_port(get_fst(redex)).x,
        //show_port(get_snd(redex)).x,
        //get_par_flag(redex),
        //show_port(get_fst(kn)).x,
        //show_port(get_snd(kn)).x);
    //}

    // Used for root redex.
    if (get_tag(a) == REF && b == ROOT) {
      rule = CALL;
    // Swaps ports if necessary.
    } else if (should_swap(a,b)) {
      swap(&a, &b);
    }

    // Dispatches interaction rule.
    bool success;
    switch (rule) {
      case LINK: success = interact_link(net, tm, a, b); break;
      case CALL: success = interact_call(net, tm, a, b); break;
      case VOID: success = interact_void(net, tm, a, b); break;
      case ERAS: success = interact_eras(net, tm, a, b); break;
      case ANNI: success = interact_anni(net, tm, a, b); break;
      case COMM: success = interact_comm(net, tm, a, b); break;
      case OPER: success = interact_oper(net, tm, a, b); break;
      case SWIT: success = interact_swit(net, tm, a, b); break;
    }

    // If error, pushes redex back.
    if (!success) {
      push_redex(tm, redex);
      return false;
    // Else, increments the interaction count.
    } else if (rule != LINK) {
      tm->itrs += 1;
    }
  }

  return true;
}

// RBag Save/Load
// --------------

// Transposes an index over a matrix.
__device__ u32 transpose(u32 idx, u32 width, u32 height) {
  u32 old_row = idx / width;
  u32 old_col = idx % width;
  u32 new_row = old_col % height;
  u32 new_col = old_col / height + old_row * (width / height);
  return new_row * width + new_col;
}


// Moves redexes from shared memory to global bag
__device__ void save_redexes(Net* net, TM *tm, u32 turn) {
  u32 idx = 0;
  u32 bag = tm->mode == SEED ? transpose(GID(), TPB, BPG) : GID();

  // Leaks low-priority redexes
  for (u32 i = 0; i < tm->rbag.lo_end; ++i) {
    Pair R = tm->rbag.lo_buf[i % RLEN];
    Port x = get_fst(R);
    Port y = get_snd(R);
    Port X = new_port(VAR, g_vars_alloc_1(net));
    Port Y = new_port(VAR, g_vars_alloc_1(net));
    vars_create(net, get_val(X), NONE);
    vars_create(net, get_val(Y), NONE);
    link_pair(net, tm, new_pair(X, x));
    link_pair(net, tm, new_pair(Y, y));
    net->g_rbag_buf_B[bag * RLEN + (idx++)] = new_pair(X, Y);
  }
  __syncthreads();
  tm->rbag.lo_end = 0;

  // Executes all high-priority redexes
  while (rbag_has_highs(&tm->rbag)) {
    Pair redex = pop_redex(tm);
    if (!interact(net, tm, redex, turn)) {
      printf("ERROR: failed to clear high-priority redexes");
    }
  }
  __syncthreads();

  #ifdef DEBUG
  if (rbag_len(&tm->rbag) > 0) printf("[%04x] ERR SAVE_REDEXES lo=%d hi=%d tot=%d\n", GID(), tm->rbag.lo_end, tm->rbag.hi_end, rbag_len(&tm->rbag));
  #endif

  // Updates global redex counter
  atomicAdd(net->g_rbag_use_B, idx);
}

// Loads redexes from global bag to shared memory
// FIXME: check if we have enuogh space for all loads
__device__ void load_redexes(Net* net, TM *tm, u32 turn) {
  u32 gid = BID() * TPB + TID();
  u32 bag = tm->mode == SEED ? transpose(GID(), TPB, BPG) : GID();
  for (u32 i = 0; i < RLEN; ++i) {
    Pair redex = atomicExch(&net->g_rbag_buf_A[bag * RLEN + i], 0);
    if (redex != 0) {
      Port a = enter(net, get_fst(redex));
      Port b = enter(net, get_snd(redex));
      #ifdef DEBUG
      if (is_local(a) || is_local(b)) printf("[%04x] ERR LOAD_REDEXES\n", turn);
      #endif
      push_redex(tm, new_pair(a, b));
    } else {
      break;
    }
  }
  __syncthreads();
}

// Kernels
// -------

// EVAL
__global__ void evaluator(GNet* gnet) {
  extern __shared__ char shared_mem[]; // 96 KB
  __shared__ Pair spawn[TPB]; // thread initialized

  // Thread Memory
  TM tm = tmem_new();

  // Net (Local-Global View)
  Net net = vnet_new(gnet, shared_mem, gnet->turn);

  // Clears shared memory
  for (u32 i = 0; i < L_NODE_LEN / TPB; ++i) {
    net.l_node_buf[i * TPB + TID()] = 0;
    net.l_vars_buf[i * TPB + TID()] = 0;
  }
  __syncthreads();

  // Sets mode
  tm.mode = gnet->mode;

  // Loads Redexes
  load_redexes(&net, &tm, gnet->turn);

  // Clears spawn buffer
  spawn[TID()] = rbag_len(&tm.rbag) > 0 ? 0xFFFFFFFFFFFFFFFF : 0;
  __syncthreads();

  // Variables
  u64 INIT = clock64(); // initial time
  u32 HASR = block_count(rbag_len(&tm.rbag) > 0);
  u32 tick = 0;
  u32 bag  = tm.mode == SEED ? transpose(GID(), TPB, BPG) : GID();
  u32 rpos = gnet->rbag_pos[bag] > 0 ? gnet->rbag_pos[bag] - gnet->rdec : gnet->rbag_pos[bag];
  u8  down = gnet->down;

  //if (BID() == 0 && gnet->turn == 0x69) {
    //printf("[%04x] ini rpos is %d | bag=%d\n", GID(), rpos, bag);
  //}

  // Aborts if empty
  if (HASR == 0) {
    return;
  }

  //if (BID() == 0 && rbag_len(&tm.rbag) > 0) {
    //Pair redex = pop_redex(&tm);
    //Pair kn = get_tag(get_snd(redex)) == CON ? node_load(&net, get_val(get_snd(redex))) : 0;
    //printf("[%04x] HAS REDEX %s ~ %s | par? %d | (%s %s)\n",
      //GID(),
      //show_port(get_fst(redex)).x,
      //show_port(get_snd(redex)).x,
      //get_par_flag(redex),
      //show_port(get_fst(kn)).x,
      //show_port(get_snd(kn)).x);
    //push_redex(&tm, redex);
  //}

  //// Display debug rbag
  //if (GID() == 0) {
    //print_rbag(&net, &tm);
    //printf("| rbag_pos = %d | mode = %d | down = %d | turn = %04x\n", gnet->rbag_pos[bag], gnet->mode, down, gnet->turn);
  //}
  //__syncthreads();

  // GROW MODE
  // ---------

  if (tm.mode == SEED || tm.mode == GROW) {
    u32 tlim = tm.mode == SEED ? min(TPB_L2,BPG_L2) : max(TPB_L2,BPG_L2);
    u32 span = 1 << (32 - __clz(TID()));

    Pair redex;

    for (u32 tick = 0; tick < tlim; ++tick) {
      u32 span = 1 << tick;
      u32 targ = TID() ^ span;

      // Attempts to spawn a thread
      if (TID() < span && spawn[targ] == 0) {
        //if (BID() == 0) {
          //if (!TID()) printf("----------------------------------------------------\n");
          //if (!TID()) printf("TIC %04x | span=%d | rlen=%d | ", tick, span, rbag_len(&tm.rbag));
          //block_print(rbag_len(&tm.rbag));
          //if (!TID()) printf("\n");
          //__syncthreads();
        //}

        // Performs some interactions until a parallel redex is found
        for (u32 i = 0; i < 64; ++i) {
          if (tm.rbag.lo_end < rpos) break;
          redex = pop_redex(&tm);
          if (redex == 0) {
            break;
          }
          // If we found a stealable redex, pass it to stealing,
          // and un-mark the redex above it, so we keep it for us.
          if (get_par_flag(redex)) {
            Pair above = pop_redex(&tm);
            if (above != 0) {
              push_redex(&tm, clr_par_flag(above));
            }
            break;
          }
          interact(&net, &tm, redex, gnet->turn);
          redex = 0;
          while (tm.rbag.hi_end > 0) {
            if (!interact(&net, &tm, pop_redex(&tm), gnet->turn)) break;
          }
        }

        // Spawn a thread
        if (redex != 0 && get_par_flag(redex)) {
          //if (BID() == 0) {
            //Pair kn = get_tag(get_snd(redex)) == CON ? node_load(&net, get_val(get_snd(redex))) : 0;
            //printf("[%04x] GIVE %s ~ %s | par? %d | (%s %s) | rbag.lo_end=%d\n", GID(), show_port(get_fst(redex)).x, show_port(get_snd(redex)).x, get_par_flag(redex), show_port(peek(&net, &tm, get_fst(kn))).x, show_port(peek(&net, &tm, get_snd(kn))).x, tm.rbag.lo_end);
          //}

          spawn[targ] = clr_par_flag(redex);
          if (!down) {
            rpos = tm.rbag.lo_end - 1;
          }
        }
      }
      __syncthreads();

      // If we've been spawned, push initial redex
      if (TID() >= span && TID() < span*2 && spawn[TID()] != 0 && spawn[TID()] != 0xFFFFFFFFFFFFFFFF) {
        //if (rbag_len(&tm.rbag) > 0) {
          //printf("[%04x] ERROR: SPAWNED BUT HAVE REDEX\n", GID());
        //}

        push_redex(&tm, atomicExch(&spawn[TID()], 0xFFFFFFFFFFFFFFFF));
        rpos = 0;
        //if (BID() == 0) printf("[%04x] TAKE %016llx\n", GID(), spawn[TID()]);
      }
      __syncthreads();

      //if (BID() == 0) {
        //if (!TID()) printf("TAC %04x | span=%d | rlen=%d | ", tick, span, rbag_len(&tm.rbag));
        //block_print(rbag_len(&tm.rbag));
        //if (!TID()) printf("\n");
        //__syncthreads();
      //}
      //__syncthreads();

      //printf("[%04x] span is %d\n", TID(), span);
      //__syncthreads();
    }

    //if (BID() == 0 && gnet->turn == 0x69) {
      //printf("[%04x] end rpos is %d | bag=%d\n", GID(), rpos, bag);
    //}

    gnet->rbag_pos[bag] = rpos;

  }

  // WORK MODE
  // ---------

  if (tm.mode == WORK) {
    u32 chkt = 0;
    u32 chka = 1;
    u32 bag  = tm.mode == SEED ? transpose(GID(), TPB, BPG) : GID();
    u32 rpos = gnet->rbag_pos[bag];
    for (tick = 0; tick < 1 << 9; ++tick) {
      if (tm.rbag.lo_end > rpos || rbag_has_highs(&tm.rbag)) {
        if (interact(&net, &tm, pop_redex(&tm), gnet->turn)) {
          while (rbag_has_highs(&tm.rbag)) {
            if (!interact(&net, &tm, pop_redex(&tm), gnet->turn)) break;
          }
        }
      }
      __syncthreads();
    }
  }
  __syncthreads();

  //u32 ITRS = block_sum(tm.itrs);
  //u32 LOOP = block_sum((u32)tick);
  //u32 RLEN = block_sum(rbag_len(&tm.rbag));
  //u32 FAIL = 0; // block_sum((u32)fail);
  //f64 TIME = (f64)(clock64() - INIT) / (f64)S;
  //f64 MIPS = (f64)ITRS / TIME / (f64)1000000.0;
  ////if (BID() >= 0 && TID() == 0) {
  //if (TID() == 0) {
    //printf("%04x:[%02x]: MODE=%d DOWN=%d ITRS=%d LOOP=%d RLEN=%d FAIL=%d TIME=%f MIPS=%.0f | %d\n",
      //gnet->turn, BID(), tm.mode, down, ITRS, LOOP, RLEN, FAIL, TIME, MIPS, 42);
  //}
  //__syncthreads();

  // Display debug rbag
  //if (BID() == 0) {
    //for (u32 i = 0; i < TPB; ++i) {
      //if (TID() == i && rbag_len(&tm.rbag) > 0) print_rbag(&net, &tm);
      //__syncthreads();
    //}
    //__syncthreads();
  //}

  // Moves rbag to global
  save_redexes(&net, &tm, gnet->turn);

  // Stores rewrites
  atomicAdd(&gnet->iadd, tm.itrs);
  atomicAdd(&gnet->leak, tm.leak);

}

// GNet Host Functions
// -------------------

void gnet_normalize(GNet* gnet) {
  // Invokes the Evaluator Kernel repeatedly
  u32 turn;
  u64 itrs = 0;
  u32 rlen = 0;
  // NORM
  for (turn = 0; turn < 0xFFFFFFFF; ++turn) {
    //printf("\e[1;1H\e[2J");
    //printf("==================================================== ");
    //printf("TURN: %04x | RLEN: %04x | ITRS: %012llu\n", turn, rlen, itrs);
    //hipDeviceSynchronize();

    evaluator<<<BPG, TPB, sizeof(LNet)>>>(gnet);
    gnet_inbetween<<<1, 1>>>(gnet);
    //hipDeviceSynchronize();

    //count_memory<<<BPG, TPB>>>(gnet);
    //hipDeviceSynchronize();

    //print_heatmap<<<1,1>>>(gnet, turn+1);
    //hipDeviceSynchronize();

    itrs = gnet_get_itrs(gnet);
    rlen = gnet_get_rlen(gnet, turn);
    if (rlen == 0) {
      //printf("Completed after %d kernel launches!\n", turn);
      break;
    }
  }
}

// Expands a REF Port.
Port gnet_expand(GNet* gnet, Port port) {
  Port old = gnet_vars_load(gnet, get_val(ROOT));
  Port got = gnet_peek(gnet, port);
  //printf("expand %s\n", show_port(got).x);
  while (get_tag(got) == REF) {
    gnet_boot_redex(gnet, new_pair(got, ROOT));
    gnet_normalize(gnet);
    got = gnet_peek(gnet, gnet_vars_load(gnet, get_val(ROOT)));
  }
  gnet_vars_create(gnet, get_val(ROOT), old);
  return got;
}

// Demos
// -----

  // stress_test 2^10 x 65536
  //static const u8 BOOK_BUF[] = {6, 0, 0, 0, 0, 0, 0, 0, 109, 97, 105, 110, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 9, 0, 0, 0, 4, 0, 0, 0, 11, 10, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 102, 117, 110, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 1, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 0, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 17, 0, 0, 0, 25, 0, 0, 0, 2, 0, 0, 0, 102, 117, 110, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 33, 0, 0, 0, 4, 0, 0, 0, 11, 0, 0, 1, 0, 0, 0, 0, 3, 0, 0, 0, 102, 117, 110, 95, 95, 67, 49, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 6, 0, 0, 0, 4, 0, 0, 0, 4, 0, 0, 0, 9, 0, 0, 128, 20, 0, 0, 0, 9, 0, 0, 128, 44, 0, 0, 0, 13, 0, 0, 0, 16, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 0, 0, 0, 0, 30, 0, 0, 0, 3, 4, 0, 0, 38, 0, 0, 0, 24, 0, 0, 0, 16, 0, 0, 0, 8, 0, 0, 0, 24, 0, 0, 0, 4, 0, 0, 0, 108, 111, 111, 112, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 1, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 0, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 11, 0, 0, 0, 41, 0, 0, 0, 5, 0, 0, 0, 108, 111, 111, 112, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 33, 0, 0, 0, 0, 0, 0, 0};

  // stress_test 2^18 x 65536
  //static const u8 BOOK_BUF[] = {6, 0, 0, 0, 0, 0, 0, 0, 109, 97, 105, 110, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 9, 0, 0, 0, 4, 0, 0, 0, 11, 18, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 102, 117, 110, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 1, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 0, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 17, 0, 0, 0, 25, 0, 0, 0, 2, 0, 0, 0, 102, 117, 110, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 33, 0, 0, 0, 4, 0, 0, 0, 11, 0, 0, 1, 0, 0, 0, 0, 3, 0, 0, 0, 102, 117, 110, 95, 95, 67, 49, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 6, 0, 0, 0, 4, 0, 0, 0, 4, 0, 0, 0, 9, 0, 0, 128, 20, 0, 0, 0, 9, 0, 0, 128, 44, 0, 0, 0, 13, 0, 0, 0, 16, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 0, 0, 0, 0, 30, 0, 0, 0, 3, 4, 0, 0, 38, 0, 0, 0, 24, 0, 0, 0, 16, 0, 0, 0, 8, 0, 0, 0, 24, 0, 0, 0, 4, 0, 0, 0, 108, 111, 111, 112, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 1, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 0, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 11, 0, 0, 0, 41, 0, 0, 0, 5, 0, 0, 0, 108, 111, 111, 112, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 33, 0, 0, 0, 0, 0, 0, 0};

  // bitonic_sort 2^20
  static const u8 BOOK_BUF[] = {19, 0, 0, 0, 0, 0, 0, 0, 109, 97, 105, 110, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 2, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 89, 0, 0, 0, 4, 0, 0, 0, 11, 18, 0, 0, 12, 0, 0, 0, 65, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 100, 111, 119, 110, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 9, 0, 0, 0, 4, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 60, 0, 0, 0, 20, 0, 0, 0, 44, 0, 0, 0, 28, 0, 0, 0, 17, 0, 0, 0, 0, 0, 0, 0, 36, 0, 0, 0, 2, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 52, 0, 0, 0, 16, 0, 0, 0, 24, 0, 0, 0, 16, 0, 0, 0, 68, 0, 0, 0, 8, 0, 0, 0, 24, 0, 0, 0, 2, 0, 0, 0, 100, 111, 119, 110, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 13, 0, 0, 0, 8, 0, 0, 0, 4, 0, 0, 0, 25, 0, 0, 128, 60, 0, 0, 0, 25, 0, 0, 128, 84, 0, 0, 0, 13, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 28, 0, 0, 0, 36, 0, 0, 0, 16, 0, 0, 0, 24, 0, 0, 0, 45, 0, 0, 0, 52, 0, 0, 0, 32, 0, 0, 0, 40, 0, 0, 0, 48, 0, 0, 0, 56, 0, 0, 0, 0, 0, 0, 0, 68, 0, 0, 0, 32, 0, 0, 0, 76, 0, 0, 0, 16, 0, 0, 0, 48, 0, 0, 0, 8, 0, 0, 0, 92, 0, 0, 0, 40, 0, 0, 0, 100, 0, 0, 0, 24, 0, 0, 0, 56, 0, 0, 0, 3, 0, 0, 0, 102, 108, 111, 119, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 9, 0, 0, 0, 4, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 60, 0, 0, 0, 20, 0, 0, 0, 44, 0, 0, 0, 28, 0, 0, 0, 33, 0, 0, 0, 0, 0, 0, 0, 36, 0, 0, 0, 2, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 52, 0, 0, 0, 16, 0, 0, 0, 24, 0, 0, 0, 16, 0, 0, 0, 68, 0, 0, 0, 8, 0, 0, 0, 24, 0, 0, 0, 4, 0, 0, 0, 102, 108, 111, 119, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 14, 0, 0, 0, 8, 0, 0, 0, 4, 0, 0, 0, 9, 0, 0, 0, 60, 0, 0, 0, 129, 0, 0, 0, 84, 0, 0, 0, 13, 0, 0, 0, 28, 0, 0, 0, 22, 0, 0, 0, 8, 0, 0, 0, 35, 1, 0, 0, 0, 0, 0, 0, 36, 0, 0, 0, 44, 0, 0, 0, 16, 0, 0, 0, 24, 0, 0, 0, 53, 0, 0, 0, 48, 0, 0, 0, 32, 0, 0, 0, 40, 0, 0, 0, 0, 0, 0, 0, 68, 0, 0, 0, 32, 0, 0, 0, 76, 0, 0, 0, 56, 0, 0, 0, 48, 0, 0, 0, 8, 0, 0, 0, 92, 0, 0, 0, 40, 0, 0, 0, 100, 0, 0, 0, 16, 0, 0, 0, 108, 0, 0, 0, 24, 0, 0, 0, 56, 0, 0, 0, 5, 0, 0, 0, 103, 101, 110, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 4, 0, 0, 0, 2, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 8, 0, 0, 0, 20, 0, 0, 0, 8, 0, 0, 0, 28, 0, 0, 0, 49, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 6, 0, 0, 0, 103, 101, 110, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 12, 0, 0, 0, 6, 0, 0, 0, 4, 0, 0, 0, 41, 0, 0, 128, 68, 0, 0, 0, 41, 0, 0, 128, 84, 0, 0, 0, 13, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 29, 0, 0, 0, 60, 0, 0, 0, 38, 0, 0, 0, 54, 0, 0, 0, 59, 2, 0, 0, 46, 0, 0, 0, 35, 1, 0, 0, 16, 0, 0, 0, 59, 2, 0, 0, 24, 0, 0, 0, 32, 0, 0, 0, 40, 0, 0, 0, 0, 0, 0, 0, 76, 0, 0, 0, 16, 0, 0, 0, 32, 0, 0, 0, 8, 0, 0, 0, 92, 0, 0, 0, 24, 0, 0, 0, 40, 0, 0, 0, 7, 0, 0, 0, 109, 97, 105, 110, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 2, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 41, 0, 0, 0, 4, 0, 0, 0, 11, 18, 0, 0, 12, 0, 0, 0, 11, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 109, 97, 105, 110, 95, 95, 67, 49, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 3, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 73, 0, 0, 0, 4, 0, 0, 0, 11, 18, 0, 0, 12, 0, 0, 0, 11, 0, 0, 0, 20, 0, 0, 0, 57, 0, 0, 0, 0, 0, 0, 0, 9, 0, 0, 0, 115, 111, 114, 116, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 9, 0, 0, 0, 4, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 60, 0, 0, 0, 20, 0, 0, 0, 44, 0, 0, 0, 28, 0, 0, 0, 81, 0, 0, 0, 0, 0, 0, 0, 36, 0, 0, 0, 2, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 52, 0, 0, 0, 16, 0, 0, 0, 24, 0, 0, 0, 16, 0, 0, 0, 68, 0, 0, 0, 8, 0, 0, 0, 24, 0, 0, 0, 10, 0, 0, 0, 115, 111, 114, 116, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 17, 0, 0, 0, 9, 0, 0, 0, 4, 0, 0, 0, 25, 0, 0, 0, 60, 0, 0, 0, 73, 0, 0, 128, 92, 0, 0, 0, 73, 0, 0, 128, 116, 0, 0, 0, 13, 0, 0, 0, 36, 0, 0, 0, 22, 0, 0, 0, 29, 0, 0, 0, 35, 1, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 16, 0, 0, 0, 44, 0, 0, 0, 52, 0, 0, 0, 24, 0, 0, 0, 32, 0, 0, 0, 40, 0, 0, 0, 48, 0, 0, 0, 0, 0, 0, 0, 68, 0, 0, 0, 40, 0, 0, 0, 76, 0, 0, 0, 84, 0, 0, 0, 48, 0, 0, 0, 56, 0, 0, 0, 64, 0, 0, 0, 8, 0, 0, 0, 100, 0, 0, 0, 11, 0, 0, 0, 108, 0, 0, 0, 24, 0, 0, 0, 56, 0, 0, 0, 16, 0, 0, 0, 124, 0, 0, 0, 11, 1, 0, 0, 132, 0, 0, 0, 32, 0, 0, 0, 64, 0, 0, 0, 11, 0, 0, 0, 115, 117, 109, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 4, 0, 0, 0, 2, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 8, 0, 0, 0, 20, 0, 0, 0, 8, 0, 0, 0, 28, 0, 0, 0, 97, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 12, 0, 0, 0, 115, 117, 109, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 10, 0, 0, 0, 6, 0, 0, 0, 4, 0, 0, 0, 89, 0, 0, 128, 36, 0, 0, 0, 89, 0, 0, 128, 68, 0, 0, 0, 13, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 28, 0, 0, 0, 32, 0, 0, 0, 16, 0, 0, 0, 24, 0, 0, 0, 0, 0, 0, 0, 44, 0, 0, 0, 16, 0, 0, 0, 54, 0, 0, 0, 3, 4, 0, 0, 62, 0, 0, 0, 40, 0, 0, 0, 32, 0, 0, 0, 8, 0, 0, 0, 76, 0, 0, 0, 24, 0, 0, 0, 40, 0, 0, 0, 13, 0, 0, 0, 115, 119, 97, 112, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 7, 0, 0, 0, 3, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 44, 0, 0, 0, 20, 0, 0, 0, 28, 0, 0, 0, 113, 0, 0, 0, 121, 0, 0, 0, 0, 0, 0, 0, 36, 0, 0, 0, 8, 0, 0, 0, 16, 0, 0, 0, 8, 0, 0, 0, 52, 0, 0, 0, 0, 0, 0, 0, 16, 0, 0, 0, 14, 0, 0, 0, 115, 119, 97, 112, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 2, 0, 0, 0, 4, 0, 0, 0, 0, 0, 0, 0, 12, 0, 0, 0, 8, 0, 0, 0, 20, 0, 0, 0, 8, 0, 0, 0, 0, 0, 0, 0, 15, 0, 0, 0, 115, 119, 97, 112, 95, 95, 67, 49, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 4, 0, 0, 0, 2, 0, 0, 0, 4, 0, 0, 0, 2, 0, 0, 0, 12, 0, 0, 0, 0, 0, 0, 0, 20, 0, 0, 0, 8, 0, 0, 0, 28, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 16, 0, 0, 0, 119, 97, 114, 112, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 9, 0, 0, 0, 4, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 52, 0, 0, 0, 20, 0, 0, 0, 28, 0, 0, 0, 137, 0, 0, 0, 145, 0, 0, 0, 0, 0, 0, 0, 36, 0, 0, 0, 8, 0, 0, 0, 44, 0, 0, 0, 16, 0, 0, 0, 24, 0, 0, 0, 16, 0, 0, 0, 60, 0, 0, 0, 8, 0, 0, 0, 68, 0, 0, 0, 0, 0, 0, 0, 24, 0, 0, 0, 17, 0, 0, 0, 119, 97, 114, 112, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 12, 0, 0, 0, 6, 0, 0, 0, 4, 0, 0, 0, 105, 0, 0, 0, 76, 0, 0, 0, 13, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 29, 0, 0, 0, 52, 0, 0, 0, 38, 0, 0, 0, 24, 0, 0, 0, 3, 15, 0, 0, 46, 0, 0, 0, 0, 0, 0, 0, 16, 0, 0, 0, 62, 0, 0, 0, 40, 0, 0, 0, 3, 18, 0, 0, 70, 0, 0, 0, 16, 0, 0, 0, 32, 0, 0, 0, 32, 0, 0, 0, 84, 0, 0, 0, 24, 0, 0, 0, 92, 0, 0, 0, 8, 0, 0, 0, 40, 0, 0, 0, 18, 0, 0, 0, 119, 97, 114, 112, 95, 95, 67, 49, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 21, 0, 0, 0, 12, 0, 0, 0, 4, 0, 0, 0, 129, 0, 0, 128, 92, 0, 0, 0, 129, 0, 0, 128, 132, 0, 0, 0, 13, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 28, 0, 0, 0, 36, 0, 0, 0, 16, 0, 0, 0, 24, 0, 0, 0, 44, 0, 0, 0, 52, 0, 0, 0, 32, 0, 0, 0, 40, 0, 0, 0, 61, 0, 0, 0, 68, 0, 0, 0, 48, 0, 0, 0, 56, 0, 0, 0, 76, 0, 0, 0, 84, 0, 0, 0, 64, 0, 0, 0, 72, 0, 0, 0, 80, 0, 0, 0, 88, 0, 0, 0, 8, 0, 0, 0, 100, 0, 0, 0, 56, 0, 0, 0, 108, 0, 0, 0, 40, 0, 0, 0, 116, 0, 0, 0, 24, 0, 0, 0, 124, 0, 0, 0, 72, 0, 0, 0, 88, 0, 0, 0, 0, 0, 0, 0, 140, 0, 0, 0, 48, 0, 0, 0, 148, 0, 0, 0, 32, 0, 0, 0, 156, 0, 0, 0, 16, 0, 0, 0, 164, 0, 0, 0, 64, 0, 0, 0, 80, 0, 0, 0};

//COMPILED_BOOK_BUF//

// Main
// ----

#ifdef IO
void do_run_io(GNet* gnet, Book* book, Port port);
#endif

extern "C" void hvm_cu(u32* book_buffer) {
  // Loads the Book
  Book* book = (Book*)malloc(sizeof(Book));
  if (book_buffer) {
    if (!book_load(book, (u32*)book_buffer)) {
      fprintf(stderr, "failed to load book\n");

      return;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(BOOK), book, sizeof(Book));
  }

  // Configures Shared Memory Size
  hipFuncSetAttribute(reinterpret_cast<const void*>(evaluator), hipFuncAttributeMaxDynamicSharedMemorySize, sizeof(LNet));

  // Creates a new GNet
  GNet* gnet = gnet_create();

  // Start the timer
  clock_t start = clock();

  // Boots root redex, to expand @main
  gnet_boot_redex(gnet, new_pair(new_port(REF, 0), ROOT));

  #ifdef IO
  do_run_io(gnet, book, ROOT);
  #else
  gnet_normalize(gnet);
  #endif

  hipDeviceSynchronize();

  // Stops the timer
  clock_t end = clock();
  double duration = ((double)(end - start)) / CLOCKS_PER_SEC;

  // Prints the result
  print_result<<<1,1>>>(gnet);

  // Reports errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch kernels. Error code: %s.\n", hipGetErrorString(err));
    if (err == hipErrorInvalidConfiguration) {
      fprintf(stderr, "Note: for now, HVM-CUDA requires a GPU with at least 128 KB of L1 cache per SM.\n");
    }
    exit(EXIT_FAILURE);
  }

  // Prints entire memdump
  //{
    //// Allocate host memory for the net
    //GNet *h_gnet = (GNet*)malloc(sizeof(GNet));

    //// Copy the net from device to host
    //hipMemcpy(h_gnet, gnet, sizeof(GNet), hipMemcpyDeviceToHost);

    //// Create a Net view of the host GNet
    //Net net;
    //net.g_node_buf = h_gnet->node_buf;
    //net.g_vars_buf = h_gnet->vars_buf;

    //// Print the net
    //print_net(&net, L_NODE_LEN, G_NODE_LEN);

    //// Free host memory
    //free(h_gnet);
  //}

  // Gets interaction count
  //hipMemcpy(&itrs, &gnet->itrs, sizeof(u64), hipMemcpyDeviceToHost);

  // Prints interactions, time and MIPS
  printf("- ITRS: %llu\n", gnet_get_itrs(gnet));
  printf("- LEAK: %llu\n", gnet_get_leak(gnet));
  printf("- TIME: %.2fs\n", duration);
  printf("- MIPS: %.2f\n", (double)gnet_get_itrs(gnet) / duration / 1000000.0);
}

#ifdef WITH_MAIN
int main() {
  hvm_cu((u32*)BOOK_BUF);
  return 0;
}
#endif
