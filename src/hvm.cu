#include "hip/hip_runtime.h"
#define INTERPRETED
#define WITHOUT_MAIN
//#define DEBUG

#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

// Integers
// --------

typedef  uint8_t u8;
typedef uint16_t u16;
typedef uint32_t u32;
typedef  int32_t i32;
typedef    float f32;
typedef   double f64;
typedef unsigned long long int u64;

#define FALSE false
#define TRUE  true

// Configuration
// -------------

// Clocks per Second
const u64 S = 2520000000;

// Threads per Block
const u32 TPB_L2 = 7;
const u32 TPB    = 1 << TPB_L2;

// Blocks per GPU
const u32 BPG_L2 = 7;
const u32 BPG    = 1 << BPG_L2;

// Threads per GPU
const u32 TPG = TPB * BPG;

//#define ALLOC_MODE SHARED
//#define ALLOC_MODE GLOBAL

// Types
// -----

// Local Types
typedef u8  Tag;  // Tag  ::= 3-bit (rounded up to u8)
typedef u32 Val;  // Val  ::= 29-bit (rounded up to u32)
typedef u32 Port; // Port ::= Tag + Val (fits a u32)
typedef u64 Pair; // Pair ::= Port + Port (fits a u64)

// Rules
typedef u8 Rule; // Rule ::= 3-bit (rounded up to 8)

// Numbs
typedef u32 Numb; // Numb ::= 29-bit (rounded up to u32)

// Tags
const Tag VAR = 0x0; // variable
const Tag REF = 0x1; // reference
const Tag ERA = 0x2; // eraser
const Tag NUM = 0x3; // number
const Tag CON = 0x4; // constructor
const Tag DUP = 0x5; // duplicator
const Tag OPR = 0x6; // operator
const Tag SWI = 0x7; // switch

// Interaction Rule Values
const Rule LINK = 0x0;
const Rule CALL = 0x1;
const Rule VOID = 0x2;
const Rule ERAS = 0x3;
const Rule ANNI = 0x4;
const Rule COMM = 0x5;
const Rule OPER = 0x6;
const Rule SWIT = 0x7;

// Constants
const Port FREE = 0x00000000;
const Port ROOT = 0xFFFFFFF8;
const Port NONE = 0xFFFFFFFF;

// Numbers
const Tag TY_SYM = 0x00;
const Tag TY_U24 = 0x01;
const Tag TY_I24 = 0x02;
const Tag TY_F24 = 0x03;
const Tag OP_ADD = 0x04;
const Tag OP_SUB = 0x05;
const Tag FP_SUB = 0x06;
const Tag OP_MUL = 0x07;
const Tag OP_DIV = 0x08;
const Tag FP_DIV = 0x09;
const Tag OP_REM = 0x0A;
const Tag FP_REM = 0x0B;
const Tag OP_EQ  = 0x0C;
const Tag OP_NEQ = 0x0D;
const Tag OP_LT  = 0x0E;
const Tag OP_GT  = 0x0F;
const Tag OP_AND = 0x10;
const Tag OP_OR  = 0x11;
const Tag OP_XOR = 0x12;
const Tag OP_SHL = 0x13;
const Tag FP_SHL = 0x14;
const Tag OP_SHR = 0x15;
const Tag FP_SHR = 0x16;

// Evaluation Modes
const u8 SEED = 0;
const u8 GROW = 1;
const u8 WORK = 2;

// Thread Redex Bag Length
const u32 RLEN = 256;

// Thread Redex Bag
// It uses the same space to store two stacks:
// - HI: a high-priotity stack, for shrinking reductions
// - LO: a low-priority stack, for growing reductions
struct RBag {
  u32  hi_end;
  Pair hi_buf[RLEN];
  u32  lo_end;
  Pair lo_buf[RLEN];
};

// Local Net
const u32 L_NODE_LEN = 0x2000;
const u32 L_VARS_LEN = 0x2000;
struct LNet {
  Pair node_buf[L_NODE_LEN];
  Port vars_buf[L_VARS_LEN];
};

// Global Net
const u32 G_NODE_LEN = 1 << 29; // max 536m nodes
const u32 G_VARS_LEN = 1 << 29; // max 536m vars
const u32 G_RBAG_LEN = TPB * BPG * RLEN * 3; // max 4m redexes
struct GNet {
  u32  rbag_use_A; // total rbag redex count (buffer A)
  u32  rbag_use_B; // total rbag redex count (buffer B)
  Pair rbag_buf_A[G_RBAG_LEN]; // global redex bag (buffer A)
  Pair rbag_buf_B[G_RBAG_LEN]; // global redex bag (buffer B)
  Pair node_buf[G_NODE_LEN]; // global node buffer
  Port vars_buf[G_VARS_LEN]; // global vars buffer
  u32  node_put[TPB*BPG];
  u32  vars_put[TPB*BPG];
  u32  rbag_pos[TPB*BPG];
  u8   mode; // evaluation mode (curr)
  u64  itrs; // interaction count
  u64  iadd; // interaction count adder
  u64  leak; // leak count
  u32  turn; // turn count
  u8   down; // are we recursing down?
  u8   rdec; // decrease rpos by 1?
};

// View Net: includes both GNet and LNet
struct Net {
  i32   l_node_dif; // delta node space
  i32   l_vars_dif; // delta vars space
  Pair *l_node_buf; // local node buffer values
  Port *l_vars_buf; // local vars buffer values
  u32  *g_rbag_use_A; // global rbag count (active buffer)
  u32  *g_rbag_use_B; // global rbag count (inactive buffer)
  Pair *g_rbag_buf_A; // global rbag values (active buffer)
  Pair *g_rbag_buf_B; // global rbag values (inactive buffer)
  Pair *g_node_buf; // global node buffer values
  Port *g_vars_buf; // global vars buffer values
  u32  *g_node_put; // next global node allocation index
  u32  *g_vars_put; // next global vars allocation index
};

// Thread Memory
struct TM {
  u32  page; // page index
  u32  nput; // node alloc index
  u32  vput; // vars alloc index
  u32  mode; // evaluation mode
  u32  itrs; // interactions
  u32  leak; // leaks
  u32  nloc[L_NODE_LEN/TPB]; // node allocs
  u32  vloc[L_NODE_LEN/TPB]; // vars allocs
  RBag rbag; // tmem redex bag
};

// Top-Level Definition
struct Def {
  char name[256];
  bool safe;
  u32  rbag_len;
  u32  node_len;
  u32  vars_len;
  Port root;
  Pair rbag_buf[L_NODE_LEN/TPB];
  Pair node_buf[L_NODE_LEN/TPB];
};

typedef struct Book Book;

// A Foreign Function
typedef struct {
  char name[256];
  Port (*func)(GNet*, Port);
} FFn;

// Book of Definitions
struct Book {
  u32 defs_len;
  Def defs_buf[0x4000];
  u32 ffns_len;
  FFn ffns_buf[0x4000];
};

// Static Book
__device__ Book BOOK;

// Debugger
// --------

struct Show {
  char x[13];
};

__device__ __host__ void put_u16(char* B, u16 val);
__device__ __host__ Show show_port(Port port);
__device__ Show show_rule(Rule rule);
__device__ void print_rbag(Net* net, TM* tm);
__device__ __host__ void print_net(Net* net, u32, u32);
__device__ void pretty_print_numb(Numb word);
__device__ void pretty_print_port(Net* net, Port port);
__device__ void pretty_print_rbag(Net* net, RBag* rbag);
__global__ void print_heatmap(GNet* gnet, u32 turn);

// Utils
// -----

__device__ __host__ f32 clamp(f32 x, f32 min, f32 max) {
  const f32 t = x < min ? min : x;
  return (t > max) ? max : t;
}

// TODO: write a time64() function that returns the time as fast as possible as a u64
static inline u64 time64() {
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  return (u64)ts.tv_sec * 1000000000ULL + (u64)ts.tv_nsec;
}

__device__ inline u32 TID() {
  return threadIdx.x;
}

__device__ inline u32 BID() {
  return blockIdx.x;
}

__device__ inline u32 GID() {
  return TID() + BID() * blockDim.x;
}

__device__ __host__ inline u32 div(u32 a, u32 b) {
  return (a + b - 1) / b;
}

__device__ u32 push_index(u32 msk, u32 idx) {
  return msk | (1U << (31 - idx));
}

__device__ u32 pop_index(u32* msk) {
  u32 idx = __clz(*msk);
  *msk &= ~(1U << (31 - idx));
  return idx;
}

// Port: Constructor and Getters
// -----------------------------

__device__ __host__ inline Port new_port(Tag tag, Val val) {
  return (val << 3) | tag;
}

__device__ __host__ inline Tag get_tag(Port port) {
  return port & 7;
}

__device__ __host__ inline Val get_val(Port port) {
  return port >> 3;
}

// Pair: Constructor and Getters
// -----------------------------

__device__ __host__ inline Pair new_pair(Port fst, Port snd) {
  return ((u64)snd << 32) | fst;
}

__device__ __host__ inline Port get_fst(Pair pair) {
  return pair & 0xFFFFFFFF;
}

__device__ __host__ inline Port get_snd(Pair pair) {
  return pair >> 32;
}

__device__ __host__ Pair set_par_flag(Pair pair) {
  Port p1 = get_fst(pair);
  Port p2 = get_snd(pair);
  if (get_tag(p1) == REF) {
    return new_pair(new_port(get_tag(p1), get_val(p1) | 0x10000000), p2);
  } else {
    return pair;
  }
}

__device__ __host__ Pair clr_par_flag(Pair pair) {
  Port p1 = get_fst(pair);
  Port p2 = get_snd(pair);
  if (get_tag(p1) == REF) {
    return new_pair(new_port(get_tag(p1), get_val(p1) & 0xFFFFFFF), p2);
  } else {
    return pair;
  }
}

__device__ __host__ bool get_par_flag(Pair pair) {
  Port p1 = get_fst(pair);
  if (get_tag(p1) == REF) {
    return (get_val(p1) >> 28) == 1;
  } else {
    return false;
  }
}

// Utils
// -----

// Swaps two ports.
__device__ __host__ inline void swap(Port *a, Port *b) {
  Port x = *a; *a = *b; *b = x;
}

// Transposes an index over a matrix.
__device__ u32 transpose(u32 idx, u32 width, u32 height) {
  u32 old_row = idx / width;
  u32 old_col = idx % width;
  u32 new_row = old_col % height;
  u32 new_col = old_col / height + old_row * (width / height);
  return new_row * width + new_col;
}

// Returns true if all 'x' are true, block-wise
__device__ __noinline__ bool block_all(bool x) {
  __shared__ bool res;
  if (TID() == 0) res = true;
  __syncthreads();
  if (!x) res = false;
  __syncthreads();
  return res;
}

// Returns true if any 'x' is true, block-wise
__device__ __noinline__ bool block_any(bool x) {
  __shared__ bool res;
  if (TID() == 0) res = false;
  __syncthreads();
  if (x) res = true;
  __syncthreads();
  return res;
}

// Returns the sum of a value, block-wise
template <typename A>
__device__ __noinline__ A block_sum(A x) {
  __shared__ A res;
  if (TID() == 0) res = 0;
  __syncthreads();
  atomicAdd(&res, x);
  __syncthreads();
  return res;
}

// Returns the sum of a boolean, block-wise
__device__ __noinline__ u32 block_count(bool x) {
  __shared__ u32 res;
  if (TID() == 0) res = 0;
  __syncthreads();
  atomicAdd(&res, x);
  __syncthreads();
  return res;
}

// Prints a 4-bit value for each thread in a block
__device__ void block_print(u32 x) {
  __shared__ u8 value[TPB];

  value[TID()] = x;
  __syncthreads();

  if (TID() == 0) {
    for (u32 i = 0; i < TPB; ++i) {
      printf("%x", min(value[i],0xF));
    }
  }
  __syncthreads();
}

// Ports / Pairs / Rules
// ---------------------

// True if this port has a pointer to a node.
__device__ __host__ inline bool is_nod(Port a) {
  return get_tag(a) >= CON;
}

// True if this port is a variable.
__device__ __host__ inline bool is_var(Port a) {
  return get_tag(a) == VAR;
}

// True if this port is a local node/var (that can leak).
__device__ __host__ inline bool is_local(Port a) {
  return (is_nod(a) || is_var(a)) && get_val(a) < L_NODE_LEN;
}

// True if this port is a global node/var (that can be leaked into).
__device__ __host__ inline bool is_global(Port a) {
  return (is_nod(a) || is_var(a)) && get_val(a) >= L_NODE_LEN;
}

// Given two tags, gets their interaction rule. Uses a u64mask lookup table.
__device__ __host__ inline Rule get_rule(Port A, Port B) {
  const u64 x = 0b0111111010110110110111101110111010110000111100001111000100000010;
  const u64 y = 0b0000110000001100000011100000110011111110111111100010111000000000;
  const u64 z = 0b1111100011111000111100001111000011000000000000000000000000000000;
  const u64 i = ((u64)get_tag(A) << 3) | (u64)get_tag(B);
  return (Rule)((x>>i&1) | (y>>i&1)<<1 | (z>>i&1)<<2);
}

// Same as above, but receiving a pair.
__device__ __host__ inline Rule get_pair_rule(Pair AB) {
  return get_rule(get_fst(AB), get_snd(AB));
}

// Should we swap ports A and B before reducing this rule?
__device__ __host__ inline bool should_swap(Port A, Port B) {
  return get_tag(B) < get_tag(A);
}
// Gets a rule's priority
__device__ __host__ inline bool is_high_priority(Rule rule) {
  return (bool)((0b00011101 >> rule) & 1);
}

// Adjusts a newly allocated port.
__device__ inline Port adjust_port(Net* net, TM* tm, Port port) {
  Tag tag = get_tag(port);
  Val val = get_val(port);
  if (is_nod(port)) return new_port(tag, tm->nloc[val]);
  if (is_var(port)) return new_port(tag, tm->vloc[val]);
  return new_port(tag, val);
}

// Adjusts a newly allocated pair.
__device__ inline Pair adjust_pair(Net* net, TM* tm, Pair pair) {
  Port p1 = adjust_port(net, tm, get_fst(pair));
  Port p2 = adjust_port(net, tm, get_snd(pair));
  return new_pair(p1, p2);
}

// Words
// -----

// Constructor and getters for SYM (operation selector)
__device__ __host__ inline Numb new_sym(u32 val) {
  return (val << 5) | TY_SYM;
}

__device__ __host__ inline u32 get_sym(Numb word) {
  return (word >> 5);
}

// Constructor and getters for U24 (unsigned 24-bit integer)
__device__ __host__ inline Numb new_u24(u32 val) {
  return (val << 5) | TY_U24;
}

__device__ __host__ inline u32 get_u24(Numb word) {
  return word >> 5;
}

// Constructor and getters for I24 (signed 24-bit integer)
__device__ __host__ inline Numb new_i24(i32 val) {
  return ((u32)val << 5) | TY_I24;
}

__device__ __host__ inline i32 get_i24(Numb word) {
  return ((i32)word) << 3 >> 8;
}

// Constructor and getters for F24 (24-bit float)
__device__ __host__ inline Numb new_f24(f32 val) {
  u32 bits = *(u32*)&val;
  u32 shifted_bits = bits >> 8;
  u32 lost_bits = bits & 0xFF;
  // round ties to even
  shifted_bits += (!isnan(val)) & ((lost_bits - ((lost_bits >> 7) & !shifted_bits)) >> 7);
  // ensure NaNs don't become infinities
  shifted_bits |= isnan(val);
  return (shifted_bits << 5) | TY_F24;
}

__device__ __host__ inline f32 get_f24(Numb word) {
  u32 bits = (word << 3) & 0xFFFFFF00;
  return *(f32*)&bits;
}

// Flip flag
__device__ __host__ inline Tag get_typ(Numb word) {
  return word & 0x1F;
}

__device__ __host__ inline bool is_num(Numb word) {
  return get_typ(word) >= TY_U24 && get_typ(word) <= TY_F24;
}

__device__ __host__ inline bool is_cast(Numb word) {
  return get_typ(word) == TY_SYM && get_sym(word) >= TY_U24 && get_sym(word) <= TY_F24;
}

// Cast a number to another type.
// The semantics are meant to spiritually resemble rust's numeric casts:
// - i24 <-> u24: is just reinterpretation of bits
// - f24  -> i24,
//   f24  -> u24: casts to the "closest" integer representing this float,
//                saturating if out of range and 0 if NaN
// - i24  -> f24,
//   u24  -> f24: casts to the "closest" float representing this integer.
__device__ __host__ inline Numb cast(Numb a, Numb b) {
  if (get_sym(a) == TY_U24 && get_typ(b) == TY_U24) return b;
  if (get_sym(a) == TY_U24 && get_typ(b) == TY_I24) {
    // reinterpret bits
    i32 val = get_i24(b);
    return new_u24(*(u32*) &val);
  }
  if (get_sym(a) == TY_U24 && get_typ(b) == TY_F24) {
    f32 val = get_f24(b);
    if (isnan(val)) {
      return new_u24(0);
    }
    return new_u24((u32) clamp(val, 0.0, 16777215));
  }

  if (get_sym(a) == TY_I24 && get_typ(b) == TY_U24) {
    // reinterpret bits
    u32 val = get_u24(b);
    return new_i24(*(i32*) &val);
  }
  if (get_sym(a) == TY_I24 && get_typ(b) == TY_I24) return b;
  if (get_sym(a) == TY_I24 && get_typ(b) == TY_F24) {
    f32 val = get_f24(b);
    if (isnan(val)) {
      return new_i24(0);
    }
    return new_i24((i32) clamp(val, -8388608.0, 8388607.0));
  }

  if (get_sym(a) == TY_F24 && get_typ(b) == TY_U24) return new_f24((f32) get_u24(b));
  if (get_sym(a) == TY_F24 && get_typ(b) == TY_I24) return new_f24((f32) get_i24(b));
  if (get_sym(a) == TY_F24 && get_typ(b) == TY_F24) return b;

  return new_u24(0);
}

// Partial application
__device__ __host__ inline Numb partial(Numb a, Numb b) {
  return (b & ~0x1F) | get_sym(a);
}

// Operate function
__device__ __host__ inline Numb operate(Numb a, Numb b) {
  Tag at = get_typ(a);
  Tag bt = get_typ(b);
  if (at == TY_SYM && bt == TY_SYM) {
    return new_u24(0);
  }
  if (is_cast(a) && is_num(b)) {
    return cast(a, b);
  }
  if (is_cast(b) && is_num(a)) {
    return cast(b, a);
  }
  if (at == TY_SYM && bt != TY_SYM) {
    return partial(a, b);
  }
  if (at != TY_SYM && bt == TY_SYM) {
    return partial(b, a);
  }
  if (at >= OP_ADD && bt >= OP_ADD) {
    return new_u24(0);
  }
  if (at < OP_ADD && bt < OP_ADD) {
    return new_u24(0);
  }
  Tag op, ty;
  Numb swp;
  if (at >= OP_ADD) {
    op = at; ty = bt;
  } else {
    op = bt; ty = at; swp = a; a = b; b = swp;
  }
  switch (ty) {
    case TY_U24: {
      u32 av = get_u24(a);
      u32 bv = get_u24(b);
      switch (op) {
        case OP_ADD: return new_u24(av + bv);
        case OP_SUB: return new_u24(av - bv);
        case FP_SUB: return new_u24(bv - av);
        case OP_MUL: return new_u24(av * bv);
        case OP_DIV: return new_u24(av / bv);
        case FP_DIV: return new_u24(bv / av);
        case OP_REM: return new_u24(av % bv);
        case FP_REM: return new_u24(bv % av);
        case OP_EQ:  return new_u24(av == bv);
        case OP_NEQ: return new_u24(av != bv);
        case OP_LT:  return new_u24(av < bv);
        case OP_GT:  return new_u24(av > bv);
        case OP_AND: return new_u24(av & bv);
        case OP_OR:  return new_u24(av | bv);
        case OP_XOR: return new_u24(av ^ bv);
        case OP_SHL: return new_u24(av << (bv & 31));
        case FP_SHL: return new_u24(bv << (av & 31));
        case OP_SHR: return new_u24(av >> (bv & 31));
        case FP_SHR: return new_u24(bv >> (av & 31));
        default:     return new_u24(0);
      }
    }
    case TY_I24: {
      i32 av = get_i24(a);
      i32 bv = get_i24(b);
      switch (op) {
        case OP_ADD: return new_i24(av + bv);
        case OP_SUB: return new_i24(av - bv);
        case FP_SUB: return new_i24(bv - av);
        case OP_MUL: return new_i24(av * bv);
        case OP_DIV: return new_i24(av / bv);
        case FP_DIV: return new_i24(bv / av);
        case OP_REM: return new_i24(av % bv);
        case FP_REM: return new_i24(bv % av);
        case OP_EQ:  return new_u24(av == bv);
        case OP_NEQ: return new_u24(av != bv);
        case OP_LT:  return new_u24(av < bv);
        case OP_GT:  return new_u24(av > bv);
        case OP_AND: return new_i24(av & bv);
        case OP_OR:  return new_i24(av | bv);
        case OP_XOR: return new_i24(av ^ bv);
        default:     return new_i24(0);
      }
    }
    case TY_F24: {
      float av = get_f24(a);
      float bv = get_f24(b);
      switch (op) {
        case OP_ADD: return new_f24(av + bv);
        case OP_SUB: return new_f24(av - bv);
        case FP_SUB: return new_f24(bv - av);
        case OP_MUL: return new_f24(av * bv);
        case OP_DIV: return new_f24(av / bv);
        case FP_DIV: return new_f24(bv / av);
        case OP_REM: return new_f24(fmodf(av, bv));
        case FP_REM: return new_f24(fmodf(bv, av));
        case OP_EQ:  return new_u24(av == bv);
        case OP_NEQ: return new_u24(av != bv);
        case OP_LT:  return new_u24(av < bv);
        case OP_GT:  return new_u24(av > bv);
        case OP_AND: return new_f24(atan2f(av, bv));
        case OP_OR:  return new_f24(logf(bv) / logf(av));
        case OP_XOR: return new_f24(powf(av, bv));
        default:     return new_f24(0);
      }
    }
    default: return new_u24(0);
  }
}

// RBag
// ----

__device__ RBag rbag_new() {
  RBag rbag;
  rbag.hi_end = 0;
  rbag.lo_end = 0;
  return rbag;
}

__device__ u32 rbag_len(RBag* rbag) {
  return rbag->hi_end + rbag->lo_end;
}

__device__ u32 rbag_has_highs(RBag* rbag) {
  return rbag->hi_end > 0;
}

__device__ void push_redex(TM* tm, Pair redex) {
  Rule rule = get_pair_rule(redex);
  if (is_high_priority(rule)) {
    tm->rbag.hi_buf[tm->rbag.hi_end++ % RLEN] = redex;
  } else {
    tm->rbag.lo_buf[tm->rbag.lo_end++ % RLEN] = redex;
  }
}

__device__ Pair pop_redex(TM* tm) {
  if (tm->rbag.hi_end > 0) {
    return tm->rbag.hi_buf[(--tm->rbag.hi_end) % RLEN];
  } else if (tm->rbag.lo_end > 0) {
    return tm->rbag.lo_buf[(--tm->rbag.lo_end) % RLEN];
  } else {
    return 0;
  }
}

// TM
// --

__device__ TM tmem_new() {
  TM tm;
  tm.rbag = rbag_new();
  tm.nput = 1;
  tm.vput = 1;
  tm.mode = SEED;
  tm.itrs = 0;
  tm.leak = 0;
  return tm;
}

// Net
// ----

__device__ Net vnet_new(GNet* gnet, void* smem, u32 turn) {
  Net net;
  net.l_node_dif   = 0;
  net.l_vars_dif   = 0;
  net.l_node_buf   = smem == NULL ? net.l_node_buf : ((LNet*)smem)->node_buf;
  net.l_vars_buf   = smem == NULL ? net.l_vars_buf : ((LNet*)smem)->vars_buf;
  net.g_rbag_use_A = turn % 2 == 0 ? &gnet->rbag_use_A : &gnet->rbag_use_B;
  net.g_rbag_use_B = turn % 2 == 0 ? &gnet->rbag_use_B : &gnet->rbag_use_A;
  net.g_rbag_buf_A = turn % 2 == 0 ? gnet->rbag_buf_A : gnet->rbag_buf_B;
  net.g_rbag_buf_B = turn % 2 == 0 ? gnet->rbag_buf_B : gnet->rbag_buf_A;
  net.g_node_buf   = gnet->node_buf;
  net.g_vars_buf   = gnet->vars_buf;
  net.g_node_put   = &gnet->node_put[GID()];
  net.g_vars_put   = &gnet->vars_put[GID()];
  return net;
}

// Stores a new node on global.
__device__ inline void node_create(Net* net, u32 loc, Pair val) {
  Pair old;
  if (loc < L_NODE_LEN) {
    net->l_node_dif += 1;
    old = atomicExch(&net->l_node_buf[loc], val);
  } else {
    old = atomicExch(&net->g_node_buf[loc], val);
  }
  #ifdef DEBUG
  if (old != 0) printf("[%04x] ERR NODE_CREATE | %04x\n", GID(), loc);
  #endif
}

// Stores a var on global.
__device__ inline void vars_create(Net* net, u32 var, Port val) {
  Port old;
  if (var < L_VARS_LEN) {
    net->l_vars_dif += 1;
    old = atomicExch(&net->l_vars_buf[var], val);
  } else {
    old = atomicExch(&net->g_vars_buf[var], val);
  }
  #ifdef DEBUG
  if (old != 0) printf("[%04x] ERR VARS_CREATE | %04x\n", GID(), var);
  #endif
}

// Reads a node from global.
__device__ __host__ inline Pair node_load(Net* net, u32 loc) {
  Pair got;
  if (loc < L_NODE_LEN) {
    got = net->l_node_buf[loc];
  } else {
    got = net->g_node_buf[loc];
  }
  return got;
}

// Reads a var from global.
__device__ __host__ inline Port vars_load(Net* net, u32 var) {
  Port got;
  if (var < L_VARS_LEN) {
    got = net->l_vars_buf[var];
  } else {
    got = net->g_vars_buf[var];
  }
  return got;
}

// Exchanges a node on global by a value. Returns old.
__device__ inline Pair node_exchange(Net* net, u32 loc, Pair val) {
  Pair got = 0;
  if (loc < L_NODE_LEN) {
    got = atomicExch(&net->l_node_buf[loc], val);
  } else {
    got = atomicExch(&net->g_node_buf[loc], val);
  }
  #ifdef DEBUG
  if (got == 0) printf("[%04x] ERR NODE_EXCHANGE | %04x\n", GID(), loc);
  #endif
  return got;
}

// Exchanges a var on global by a value. Returns old.
__device__ inline Port vars_exchange(Net* net, u32 var, Port val) {
  Port got = 0;
  if (var < L_VARS_LEN) {
    got = atomicExch(&net->l_vars_buf[var], val);
  } else {
    got = atomicExch(&net->g_vars_buf[var], val);
  }
  #ifdef DEBUG
  if (got == 0) printf("[%04x] ERR VARS_EXCHANGE | %04x\n", GID(), var);
  #endif
  return got;
}

// Takes a node.
__device__ inline Pair node_take(Net* net, u32 loc) {
  Pair got = 0;
  if (loc < L_NODE_LEN) {
    net->l_node_dif -= 1;
    got = atomicExch(&net->l_node_buf[loc], 0);
  } else {
    got = atomicExch(&net->g_node_buf[loc], 0);
  }
  #ifdef DEBUG
  if (got == 0) printf("[%04x] ERR NODE_TAKE | %04x\n", GID(), loc);
  #endif
  return got;
}

// Takes a var.
__device__ inline Port vars_take(Net* net, u32 var) {
  Port got = 0;
  if (var < L_VARS_LEN) {
    net->l_vars_dif -= 1;
    got = atomicExch(&net->l_vars_buf[var], 0);
  } else {
    got = atomicExch(&net->g_vars_buf[var], 0);
  }
  #ifdef DEBUG
  if (got == 0) printf("[%04x] ERR VARS_TAKE | %04x\n", GID(), var);
  #endif
  return got;
}

// Allocator
// ---------

template <typename A>
__device__ u32 g_alloc_1(Net* net, TM* tm, u32* g_put, A* g_buf) {
  u32 lps = 0;
  while (true) {
    u32 lc = GID()*(G_NODE_LEN/TPG) + (*g_put%(G_NODE_LEN/TPG));
    A elem = g_buf[lc];
    *g_put += 1;
    if (lc >= L_NODE_LEN && elem == 0) {
      return lc;
    }
    if (++lps >= G_NODE_LEN/TPG) printf("OOM\n"); // FIXME: remove
    //assert(++lps < G_NODE_LEN/TPG); // FIXME: enable?
  }
}

template <typename A>
__device__ u32 g_alloc(Net* net, TM* tm, u32* ret, u32* g_put, A* g_buf, u32 num) {
  u32 got = 0;
  u32 lps = 0;
  while (got < num) {
    u32 lc = GID()*(G_NODE_LEN/TPG) + (*g_put%(G_NODE_LEN/TPG));
    A elem = g_buf[lc];
    *g_put += 1;
    if (lc >= L_NODE_LEN && elem == 0) {
      ret[got++] = lc;
    }
    if (++lps >= G_NODE_LEN/TPG) printf("OOM\n"); // FIXME: remove
    //assert(++lps < G_NODE_LEN/TPG); // FIXME: enable?
  }
  return got;

}

template <typename A>
__device__ u32 l_alloc(Net* net, TM* tm, u32* ret, u32* l_put, A* l_buf, u32 num) {
  u32 got = 0;
  u32 lps = 0;
  while (got < num) {
    u32 lc = ((*l_put)++ * TPB) % L_NODE_LEN + TID();
    A elem = l_buf[lc];
    if (++lps >= L_NODE_LEN/TPB) {
      break;
    }
    if (lc > 0 && elem == 0) {
      ret[got++] = lc;
    }
  }
  return got;
}

template <typename A>
__device__ u32 l_alloc_1(Net* net, TM* tm, u32* ret, u32* l_put, A* l_buf, u32* lps) {
  u32 got = 0;
  while (true) {
    u32 lc = ((*l_put)++ * TPB) % L_NODE_LEN + TID();
    A elem = l_buf[lc];
    if (++(*lps) >= L_NODE_LEN/TPB) {
      break;
    }
    if (lc > 0 && elem == 0) {
      return lc;
    }
  }
  return got;
}

__device__ u32 g_node_alloc_1(Net* net, TM* tm) {
  return g_alloc_1(net, tm, net->g_node_put, net->g_node_buf);
}

__device__ u32 g_vars_alloc_1(Net* net, TM* tm) {
  return g_alloc_1(net, tm, net->g_vars_put, net->g_vars_buf);
}

__device__ u32 g_node_alloc(Net* net, TM* tm, u32 num) {
  return g_alloc(net, tm, tm->nloc, net->g_node_put, net->g_node_buf, num);
}

__device__ u32 g_vars_alloc(Net* net, TM* tm, u32 num) {
  return g_alloc(net, tm, tm->vloc, net->g_vars_put, net->g_vars_buf, num);
}

__device__ u32 l_node_alloc(Net* net, TM* tm, u32 num) {
  return l_alloc(net, tm, tm->nloc, &tm->nput, net->l_node_buf, num);
}

__device__ u32 l_vars_alloc(Net* net, TM* tm, u32 num) {
  return l_alloc(net, tm, tm->vloc, &tm->vput, net->l_vars_buf, num);
}

__device__ u32 l_node_alloc_1(Net* net, TM* tm, u32* lps) {
  return l_alloc_1(net, tm, tm->nloc, &tm->nput, net->l_node_buf, lps);
}

__device__ u32 l_vars_alloc_1(Net* net, TM* tm, u32* lps) {
  return l_alloc_1(net, tm, tm->vloc, &tm->vput, net->l_vars_buf, lps);
}

__device__ u32 node_alloc_1(Net* net, TM* tm, u32* lps) {
  if (tm->mode != WORK) {
    return g_node_alloc_1(net, tm);
  } else {
    return l_node_alloc_1(net, tm, lps);
  }
}

__device__ u32 vars_alloc_1(Net* net, TM* tm, u32* lps) {
  if (tm->mode != WORK) {
    return g_vars_alloc_1(net, tm);
  } else {
    return l_vars_alloc_1(net, tm, lps);
  }
}

// Linking
// -------

// Finds a variable's value.
__device__ inline Port peek(Net* net, TM* tm, Port var) {
  while (get_tag(var) == VAR) {
    Port val = vars_load(net, get_val(var));
    if (val == NONE) break;
    if (val == 0) break;
    var = val;
  }
  return var;
}

// Finds a variable's value.
__device__ inline Port enter(Net* net, TM* tm, Port var) {
  u32 lps = 0;
  Port init = var;
  // While `B` is VAR: extend it (as an optimization)
  while (get_tag(var) == VAR) {
    // Takes the current `var` substitution as `val`
    Port val = vars_exchange(net, get_val(var), NONE);
    // If there was no `val`, stop, as there is no extension
    if (val == NONE) {
      break;
    }
    // Sanity check: if global A is unfilled, stop
    if (val == 0) {
      break;
    }
    // Otherwise, delete `B` (we own both) and continue
    vars_take(net, get_val(var));
    //if (++lps > 65536) printf("[%04x] BUG A | init=%s var=%s val=%s\n", GID(), show_port(init).x, show_port(var).x, show_port(val).x);
    var = val;
  }
  return var;
}

// Atomically Links `A ~ B`.
__device__ void link(Net* net, TM* tm, Port A, Port B) {
  Port INI_A = A;
  Port INI_B = B;

  u32 lps = 0;

  // Attempts to directionally point `A ~> B`
  while (true) {

    // If `A` is NODE: swap `A` and `B`, and continue
    if (get_tag(A) != VAR && get_tag(B) == VAR) {
      Port X = A; A = B; B = X;
    }

    // If `A` is NODE: create the `A ~ B` redex
    if (get_tag(A) != VAR) {
      //printf("[%04x] new redex A %s ~ %s\n", GID(), show_port(A).x, show_port(B).x);
      push_redex(tm, new_pair(A, B)); // TODO: move global ports to local
      break;
    }

    // While `B` is VAR: extend it (as an optimization)
    B = enter(net, tm, B);

    // Since `A` is VAR: point `A ~> B`.
    if (true) {
      // If B would leak...
      if (is_global(A) && is_local(B)) {
        // If B is a var, just swap it
        if (is_var(B)) {
          Port X = A; A = B; B = X;
          continue;
        }
        // If B is a nod, create a leak interaction
        if (is_nod(B)) {
          //if (!TID()) printf("[%04x] NODE LEAK %s ~ %s\n", GID(), show_port(A).x, show_port(B).x);
          push_redex(tm, new_pair(A, B));
          break;
        }
      }

      // Sanity check: if global A is unfilled, delay this link
      if (is_global(A) && vars_load(net, get_val(A)) == 0) {
        push_redex(tm, new_pair(A, B));
        break;
      }

      // Stores `A -> B`, taking the current `A` subst as `A'`
      Port A_ = vars_exchange(net, get_val(A), B);

      // If there was no `A'`, stop, as we lost B's ownership
      if (A_ == NONE) {
        break;
      }

      #ifdef DEBUG
      if (A_ == 0) printf("[%04x] ERR LINK %s ~ %s | %s ~ %s\n", GID(), show_port(INI_A).x, show_port(INI_B).x, show_port(A).x, show_port(B).x);
      #endif

      // Otherwise, delete `A` (we own both) and link `A' ~ B`
      vars_take(net, get_val(A));
      A = A_;
    }
  }
}

// Links `A ~ B` (as a pair).
__device__ void link_pair(Net* net, TM* tm, Pair AB) {
  link(net, tm, get_fst(AB), get_snd(AB));
}

// Resources
// ---------

// Gets the necessary resources for an interaction.
__device__ bool get_resources(Net* net, TM* tm, u8 need_rbag, u8 need_node, u8 need_vars) {
  u32 got_rbag = min(RLEN - tm->rbag.lo_end, RLEN - tm->rbag.hi_end);
  u32 got_node;
  u32 got_vars;
  if (tm->mode != WORK) {
    got_node = g_node_alloc(net, tm, need_node);
    got_vars = g_vars_alloc(net, tm, need_vars);
  } else {
    got_node = l_node_alloc(net, tm, need_node);
    got_vars = l_vars_alloc(net, tm, need_vars);
  }
  return got_rbag >= need_rbag && got_node >= need_node && got_vars >= need_vars;
}

// Interactions
// ------------

// The Link Interaction.
__device__ bool interact_link(Net* net, TM* tm, Port a, Port b) {
  // If A is a global var and B is a local node, leak it:
  // ^A ~ (b1 b2)
  // ------------- LEAK-NODE
  // ^X ~ b1
  // ^Y ~ b2
  // ^A ~ ^(^X ^Y)
  if (is_global(a) && is_nod(b) && is_local(b)) {
    // Allocates needed nodes and vars.
    if (!get_resources(net, tm, 3, 0, 0)) {
      return false;
    }

    tm->leak += 1;

    // Loads ports.
    Pair l_b  = node_take(net, get_val(b));
    Port l_b1 = enter(net, tm, get_fst(l_b));
    Port l_b2 = enter(net, tm, get_snd(l_b));

    // Leaks port 1.
    Port g_b1;
    if (is_local(l_b1)) {
      g_b1 = new_port(VAR, g_vars_alloc_1(net, tm));
      vars_create(net, get_val(g_b1), NONE);
      link_pair(net, tm, new_pair(g_b1, l_b1));
    } else {
      g_b1 = l_b1;
    }

    // Leaks port 2.
    Port g_b2;
    if (is_local(l_b2)) {
      g_b2 = new_port(VAR, g_vars_alloc_1(net, tm));
      vars_create(net, get_val(g_b2), NONE);
      link_pair(net, tm, new_pair(g_b2, l_b2));
    } else {
      g_b2 = l_b2;
    }

    // Leaks node.
    Port g_b = new_port(get_tag(b), g_node_alloc_1(net, tm));
    node_create(net, get_val(g_b), new_pair(g_b1, g_b2));
    link_pair(net, tm, new_pair(a, g_b));

    return true;

  // Otherwise, just perform a normal link.
  } else {
    // Allocates needed nodes and vars.
    if (!get_resources(net, tm, 1, 0, 0)) {
      return false;
    }

    link_pair(net, tm, new_pair(a, b));
  }

  return true;
}

// Declared here for use in call interactions.
static inline bool interact_eras(Net* net, TM* tm, Port a, Port b);

// The Call Interaction.
#ifdef COMPILED
///COMPILED_INTERACT_CALL///
#else
__device__ bool interact_eras(Net* net, TM* tm, Port a, Port b);
__device__ bool interact_call(Net* net, TM* tm, Port a, Port b) {
  // Loads Definition.
  u32 fid  = get_val(a) & 0xFFFFFFF;
  Def* def = &BOOK.defs_buf[fid];

  // Copy Optimization.
  if (def->safe && get_tag(b) == DUP) {
    return interact_eras(net, tm, a, b);
  }

  // Allocates needed nodes and vars.
  if (!get_resources(net, tm, def->rbag_len + 1, def->node_len, def->vars_len)) {
    return false;
  }

  // Stores new vars.
  for (u32 i = 0; i < def->vars_len; ++i) {
    vars_create(net, tm->vloc[i], NONE);
  }

  // Stores new nodes.
  for (u32 i = 0; i < def->node_len; ++i) {
    node_create(net, tm->nloc[i], adjust_pair(net, tm, def->node_buf[i]));
  }

  // Links.
  for (u32 i = 0; i < def->rbag_len; ++i) {
    link_pair(net, tm, adjust_pair(net, tm, def->rbag_buf[i]));
  }
  link_pair(net, tm, new_pair(adjust_port(net, tm, def->root), b));

  return true;
}
#endif

// The Void Interaction.
__device__ bool interact_void(Net* net, TM* tm, Port a, Port b) {
  return true;
}

// The Eras Interaction.
__device__ bool interact_eras(Net* net, TM* tm, Port a, Port b) {
  // Allocates needed nodes and vars.
  if (!get_resources(net, tm, 2, 0, 0)) {
    return false;
  }

  // Loads ports.
  Pair B  = node_take(net, get_val(b));
  Port B1 = get_fst(B);
  Port B2 = get_snd(B);

  // Links.
  link_pair(net, tm, new_pair(a, B1));
  link_pair(net, tm, new_pair(a, B2));

  return true;
}

// The Anni Interaction.
__device__ bool interact_anni(Net* net, TM* tm, Port a, Port b) {
  // Allocates needed nodes and vars.
  if (!get_resources(net, tm, 2, 0, 0)) {
    return false;
  }

  // Loads ports.
  Pair A  = node_take(net, get_val(a));
  Port A1 = get_fst(A);
  Port A2 = get_snd(A);
  Pair B  = node_take(net, get_val(b));
  Port B1 = get_fst(B);
  Port B2 = get_snd(B);

  // Links.
  link_pair(net, tm, new_pair(A1, B1));
  link_pair(net, tm, new_pair(A2, B2));

  return true;
}

// The Comm Interaction.
__device__ bool interact_comm(Net* net, TM* tm, Port a, Port b) {
  // Allocates needed nodes and vars.
  if (!get_resources(net, tm, 4, 4, 4)) {
    return false;
  }

  // Loads ports.
  Pair A  = node_take(net, get_val(a));
  Port A1 = get_fst(A);
  Port A2 = get_snd(A);
  Pair B  = node_take(net, get_val(b));
  Port B1 = get_fst(B);
  Port B2 = get_snd(B);

  // Stores new vars.
  vars_create(net, tm->vloc[0], NONE);
  vars_create(net, tm->vloc[1], NONE);
  vars_create(net, tm->vloc[2], NONE);
  vars_create(net, tm->vloc[3], NONE);

  // Stores new nodes.
  node_create(net, tm->nloc[0], new_pair(new_port(VAR, tm->vloc[0]), new_port(VAR, tm->vloc[1])));
  node_create(net, tm->nloc[1], new_pair(new_port(VAR, tm->vloc[2]), new_port(VAR, tm->vloc[3])));
  node_create(net, tm->nloc[2], new_pair(new_port(VAR, tm->vloc[0]), new_port(VAR, tm->vloc[2])));
  node_create(net, tm->nloc[3], new_pair(new_port(VAR, tm->vloc[1]), new_port(VAR, tm->vloc[3])));

  // Links.
  link_pair(net, tm, new_pair(new_port(get_tag(b), tm->nloc[0]), A1));
  link_pair(net, tm, new_pair(new_port(get_tag(b), tm->nloc[1]), A2));
  link_pair(net, tm, new_pair(new_port(get_tag(a), tm->nloc[2]), B1));
  link_pair(net, tm, new_pair(new_port(get_tag(a), tm->nloc[3]), B2));

  return true;
}

// The Oper Interaction.
__device__ bool interact_oper(Net* net, TM* tm, Port a, Port b) {
  // Allocates needed nodes and vars.
  if (!get_resources(net, tm, 1, 1, 0)) {
    return false;
  }

  // Loads ports.
  Val  av = get_val(a);
  Pair B  = node_take(net, get_val(b));
  Port B1 = get_fst(B);
  Port B2 = enter(net, tm, get_snd(B));

  // Performs operation.
  if (get_tag(B1) == NUM) {
    Val  bv = get_val(B1);
    Numb cv = operate(av, bv);
    link_pair(net, tm, new_pair(new_port(NUM, cv), B2));
  } else {
    node_create(net, tm->nloc[0], new_pair(a, B2));
    link_pair(net, tm, new_pair(B1, new_port(OPR, tm->nloc[0])));
  }

  return true;
}

// The Swit Interaction.
__device__ bool interact_swit(Net* net, TM* tm, Port a, Port b) {
  // Allocates needed nodes and vars.
  if (!get_resources(net, tm, 1, 2, 0)) {
    return false;
  }

  // Loads ports.
  u32  av = get_u24(get_val(a));
  Pair B  = node_take(net, get_val(b));
  Port B1 = get_fst(B);
  Port B2 = get_snd(B);

  // Stores new nodes.
  if (av == 0) {
    node_create(net, tm->nloc[0], new_pair(B2, new_port(ERA,0)));
    link_pair(net, tm, new_pair(new_port(CON, tm->nloc[0]), B1));
  } else {
    node_create(net, tm->nloc[0], new_pair(new_port(ERA,0), new_port(CON, tm->nloc[1])));
    node_create(net, tm->nloc[1], new_pair(new_port(NUM, new_u24(av-1)), B2));
    link_pair(net, tm, new_pair(new_port(CON, tm->nloc[0]), B1));
  }

  return true;
}

// Pops a local redex and performs a single interaction.
__device__ bool interact(Net* net, TM* tm, Pair redex, u32 turn) {
  // Gets redex ports A and B.
  Port a = get_fst(redex);
  Port b = get_snd(redex);

  // Gets the rule type.
  Rule rule = get_rule(a, b);

  // If there is no redex, stop.
  if (redex != 0) {
    //if (GID() == 0 && turn == 0x201) {
      //Pair kn = get_tag(b) == CON ? node_load(net, get_val(b)) : 0;
      //printf("%04x:[%04x] REDUCE %s ~ %s | par? %d | (%s %s)\n",
        //turn, GID(),
        //show_port(get_fst(redex)).x,
        //show_port(get_snd(redex)).x,
        //get_par_flag(redex),
        //show_port(get_fst(kn)).x,
        //show_port(get_snd(kn)).x);
    //}

    // Used for root redex.
    if (get_tag(a) == REF && b == ROOT) {
      rule = CALL;
    // Swaps ports if necessary.
    } else if (should_swap(a,b)) {
      swap(&a, &b);
    }

    // Dispatches interaction rule.
    bool success;
    switch (rule) {
      case LINK: success = interact_link(net, tm, a, b); break;
      case CALL: success = interact_call(net, tm, a, b); break;
      case VOID: success = interact_void(net, tm, a, b); break;
      case ERAS: success = interact_eras(net, tm, a, b); break;
      case ANNI: success = interact_anni(net, tm, a, b); break;
      case COMM: success = interact_comm(net, tm, a, b); break;
      case OPER: success = interact_oper(net, tm, a, b); break;
      case SWIT: success = interact_swit(net, tm, a, b); break;
    }

    // If error, pushes redex back.
    if (!success) {
      push_redex(tm, redex);
      return false;
    // Else, increments the interaction count.
    } else if (rule != LINK) {
      tm->itrs += 1;
    }
  }

  return true;
}

// RBag Save/Load
// --------------

// Moves redexes from shared memory to global bag
__device__ void save_redexes(Net* net, TM *tm, u32 turn) {
  u32 idx = 0;
  u32 bag = tm->mode == SEED ? transpose(GID(), TPB, BPG) : GID();

  // Leaks low-priority redexes
  for (u32 i = 0; i < tm->rbag.lo_end; ++i) {
    Pair R = tm->rbag.lo_buf[i % RLEN];
    Port x = get_fst(R);
    Port y = get_snd(R);
    Port X = new_port(VAR, g_vars_alloc_1(net, tm));
    Port Y = new_port(VAR, g_vars_alloc_1(net, tm));
    vars_create(net, get_val(X), NONE);
    vars_create(net, get_val(Y), NONE);
    link_pair(net, tm, new_pair(X, x));
    link_pair(net, tm, new_pair(Y, y));
    net->g_rbag_buf_B[bag * RLEN + (idx++)] = new_pair(X, Y);
  }
  __syncthreads();
  tm->rbag.lo_end = 0;

  // Executes all high-priority redexes
  while (rbag_has_highs(&tm->rbag)) {
    Pair redex = pop_redex(tm);
    if (!interact(net, tm, redex, turn)) {
      printf("ERROR: failed to clear high-priority redexes");
    }
  }
  __syncthreads();

  #ifdef DEBUG
  if (rbag_len(&tm->rbag) > 0) printf("[%04x] ERR SAVE_REDEXES lo=%d hi=%d tot=%d\n", GID(), tm->rbag.lo_end, tm->rbag.hi_end, rbag_len(&tm->rbag));
  #endif

  // Updates global redex counter
  atomicAdd(net->g_rbag_use_B, idx);
}

// Loads redexes from global bag to shared memory
// FIXME: check if we have enuogh space for all loads
__device__ void load_redexes(Net* net, TM *tm, u32 turn) {
  u32 gid = BID() * TPB + TID();
  u32 bag = tm->mode == SEED ? transpose(GID(), TPB, BPG) : GID();
  for (u32 i = 0; i < RLEN; ++i) {
    Pair redex = atomicExch(&net->g_rbag_buf_A[bag * RLEN + i], 0);
    if (redex != 0) {
      Port a = enter(net, tm, get_fst(redex));
      Port b = enter(net, tm, get_snd(redex));
      #ifdef DEBUG
      if (is_local(a) || is_local(b)) printf("[%04x] ERR LOAD_REDEXES\n", turn);
      #endif
      push_redex(tm, new_pair(a, b));
    } else {
      break;
    }
  }
  __syncthreads();
}

// Kernels
// -------

// Sets the initial redex.
__global__ void boot_redex(GNet* gnet, Pair redex) {
  // Creates root variable.
  gnet->vars_buf[get_val(ROOT)] = NONE;
  // Creates root redex.
  if (gnet->turn % 2 == 0) {
    gnet->rbag_buf_A[0] = redex;
  } else {
    gnet->rbag_buf_B[0] = redex;
  }
}

// Creates a node.
__global__ void make_node(GNet* gnet, Tag tag, Port fst, Port snd, Port* ret) {
  if (GID() == 0) {
    TM tm;
    Net net = vnet_new(gnet, NULL, gnet->turn);
    u32 loc = g_node_alloc_1(&net, &tm);
    node_create(&net, loc, new_pair(fst, snd));
    *ret = new_port(tag, loc);
  }
}

__global__ void inbetween(GNet* gnet) {
  // Clears rbag use counter
  if (gnet->turn % 2 == 0) {
    gnet->rbag_use_A = 0;
  } else {
    gnet->rbag_use_B = 0;
  }

  // Increments gnet turn
  gnet->turn += 1;

  // Increments interaction counter
  gnet->itrs += gnet->iadd;

  // Resets the rdec variable
  gnet->rdec = 0;

  // Moves to next mode
  if (!gnet->down) {
    gnet->mode = min(gnet->mode + 1, WORK);
  }

  // If no work was done...
  if (gnet->iadd == 0) {
    // If on seed mode, go up to GROW mode
    if (gnet->mode == SEED) {
      gnet->mode = GROW;
      gnet->down = 0;
    // Otherwise, go down to SEED mode
    } else {
      gnet->mode = SEED;
      gnet->down = 1;
      gnet->rdec = 1; // peel one rpos
    }
    //printf(">> CHANGE MODE TO %d | %d <<\n", gnet->mode, gnet->down);
  }

  // Reset interaction adder
  gnet->iadd = 0;
}

// EVAL
__global__ void evaluator(GNet* gnet) {
  extern __shared__ char shared_mem[]; // 96 KB
  __shared__ Pair spawn[TPB]; // thread initialized

  // Thread Memory
  TM tm = tmem_new();

  // Net (Local-Global View)
  Net net = vnet_new(gnet, shared_mem, gnet->turn);

  // Clears shared memory
  for (u32 i = 0; i < L_NODE_LEN / TPB; ++i) {
    net.l_node_buf[i * TPB + TID()] = 0;
    net.l_vars_buf[i * TPB + TID()] = 0;
  }
  __syncthreads();

  // Sets mode
  tm.mode = gnet->mode;

  // Loads Redexes
  load_redexes(&net, &tm, gnet->turn);

  // Clears spawn buffer
  spawn[TID()] = rbag_len(&tm.rbag) > 0 ? 0xFFFFFFFFFFFFFFFF : 0;
  __syncthreads();

  // Variables
  u64 INIT = clock64(); // initial time
  u32 HASR = block_count(rbag_len(&tm.rbag) > 0);
  u32 tick = 0;
  u32 bag  = tm.mode == SEED ? transpose(GID(), TPB, BPG) : GID();
  u32 rpos = gnet->rbag_pos[bag] > 0 ? gnet->rbag_pos[bag] - gnet->rdec : gnet->rbag_pos[bag];
  u8  down = gnet->down;

  //if (BID() == 0 && gnet->turn == 0x69) {
    //printf("[%04x] ini rpos is %d | bag=%d\n", GID(), rpos, bag);
  //}

  // Aborts if empty
  if (HASR == 0) {
    return;
  }

  //if (BID() == 0 && rbag_len(&tm.rbag) > 0) {
    //Pair redex = pop_redex(&tm);
    //Pair kn = get_tag(get_snd(redex)) == CON ? node_load(&net, get_val(get_snd(redex))) : 0;
    //printf("[%04x] HAS REDEX %s ~ %s | par? %d | (%s %s)\n",
      //GID(),
      //show_port(get_fst(redex)).x,
      //show_port(get_snd(redex)).x,
      //get_par_flag(redex),
      //show_port(get_fst(kn)).x,
      //show_port(get_snd(kn)).x);
    //push_redex(&tm, redex);
  //}

  //// Display debug rbag
  //if (GID() == 0) {
    //print_rbag(&net, &tm);
    //printf("| rbag_pos = %d | mode = %d | down = %d | turn = %04x\n", gnet->rbag_pos[bag], gnet->mode, down, gnet->turn);
  //}
  //__syncthreads();

  // GROW MODE
  // ---------

  if (tm.mode == SEED || tm.mode == GROW) {
    u32 tlim = tm.mode == SEED ? min(TPB_L2,BPG_L2) : max(TPB_L2,BPG_L2);
    u32 span = 1 << (32 - __clz(TID()));

    Pair redex;

    for (u32 tick = 0; tick < tlim; ++tick) {
      u32 span = 1 << tick;
      u32 targ = TID() ^ span;

      // Attempts to spawn a thread
      if (TID() < span && spawn[targ] == 0) {
        //if (BID() == 0) {
          //if (!TID()) printf("----------------------------------------------------\n");
          //if (!TID()) printf("TIC %04x | span=%d | rlen=%d | ", tick, span, rbag_len(&tm.rbag));
          //block_print(rbag_len(&tm.rbag));
          //if (!TID()) printf("\n");
          //__syncthreads();
        //}

        // Performs some interactions until a parallel redex is found
        for (u32 i = 0; i < 64; ++i) {
          if (tm.rbag.lo_end < rpos) break;
          redex = pop_redex(&tm);
          if (redex == 0) {
            break;
          }
          // If we found a stealable redex, pass it to stealing,
          // and un-mark the redex above it, so we keep it for us.
          if (get_par_flag(redex)) {
            Pair above = pop_redex(&tm);
            if (above != 0) {
              push_redex(&tm, clr_par_flag(above));
            }
            break;
          }
          interact(&net, &tm, redex, gnet->turn);
          redex = 0;
          while (tm.rbag.hi_end > 0) {
            if (!interact(&net, &tm, pop_redex(&tm), gnet->turn)) break;
          }
        }

        // Spawn a thread
        if (redex != 0 && get_par_flag(redex)) {
          //if (BID() == 0) {
            //Pair kn = get_tag(get_snd(redex)) == CON ? node_load(&net, get_val(get_snd(redex))) : 0;
            //printf("[%04x] GIVE %s ~ %s | par? %d | (%s %s) | rbag.lo_end=%d\n", GID(), show_port(get_fst(redex)).x, show_port(get_snd(redex)).x, get_par_flag(redex), show_port(peek(&net, &tm, get_fst(kn))).x, show_port(peek(&net, &tm, get_snd(kn))).x, tm.rbag.lo_end);
          //}

          spawn[targ] = clr_par_flag(redex);
          if (!down) {
            rpos = tm.rbag.lo_end - 1;
          }
        }
      }
      __syncthreads();

      // If we've been spawned, push initial redex
      if (TID() >= span && TID() < span*2 && spawn[TID()] != 0 && spawn[TID()] != 0xFFFFFFFFFFFFFFFF) {
        //if (rbag_len(&tm.rbag) > 0) {
          //printf("[%04x] ERROR: SPAWNED BUT HAVE REDEX\n", GID());
        //}

        push_redex(&tm, atomicExch(&spawn[TID()], 0xFFFFFFFFFFFFFFFF));
        rpos = 0;
        //if (BID() == 0) printf("[%04x] TAKE %016llx\n", GID(), spawn[TID()]);
      }
      __syncthreads();

      //if (BID() == 0) {
        //if (!TID()) printf("TAC %04x | span=%d | rlen=%d | ", tick, span, rbag_len(&tm.rbag));
        //block_print(rbag_len(&tm.rbag));
        //if (!TID()) printf("\n");
        //__syncthreads();
      //}
      //__syncthreads();

      //printf("[%04x] span is %d\n", TID(), span);
      //__syncthreads();
    }

    //if (BID() == 0 && gnet->turn == 0x69) {
      //printf("[%04x] end rpos is %d | bag=%d\n", GID(), rpos, bag);
    //}

    gnet->rbag_pos[bag] = rpos;

  }

  // WORK MODE
  // ---------

  if (tm.mode == WORK) {
    u32 chkt = 0;
    u32 chka = 1;
    u32 bag  = tm.mode == SEED ? transpose(GID(), TPB, BPG) : GID();
    u32 rpos = gnet->rbag_pos[bag];
    for (tick = 0; tick < 1 << 9; ++tick) {
      if (tm.rbag.lo_end > rpos) {
        if (interact(&net, &tm, pop_redex(&tm), gnet->turn)) {
          while (rbag_has_highs(&tm.rbag)) {
            if (!interact(&net, &tm, pop_redex(&tm), gnet->turn)) break;
          }
        }
      }
      __syncthreads();
    }
  }
  __syncthreads();

  //u32 ITRS = block_sum(tm.itrs);
  //u32 LOOP = block_sum((u32)tick);
  //u32 RLEN = block_sum(rbag_len(&tm.rbag));
  //u32 FAIL = 0; // block_sum((u32)fail);
  //f64 TIME = (f64)(clock64() - INIT) / (f64)S;
  //f64 MIPS = (f64)ITRS / TIME / (f64)1000000.0;
  ////if (BID() >= 0 && TID() == 0) {
  //if (TID() == 0) {
    //printf("%04x:[%02x]: MODE=%d DOWN=%d ITRS=%d LOOP=%d RLEN=%d FAIL=%d TIME=%f MIPS=%.0f | %d\n",
      //gnet->turn, BID(), tm.mode, down, ITRS, LOOP, RLEN, FAIL, TIME, MIPS, 42);
  //}
  //__syncthreads();

  // Display debug rbag
  //if (BID() == 0) {
    //for (u32 i = 0; i < TPB; ++i) {
      //if (TID() == i && rbag_len(&tm.rbag) > 0) print_rbag(&net, &tm);
      //__syncthreads();
    //}
    //__syncthreads();
  //}

  // Moves rbag to global
  save_redexes(&net, &tm, gnet->turn);

  // Stores rewrites
  atomicAdd(&gnet->iadd, tm.itrs);
  atomicAdd(&gnet->leak, tm.leak);

}

// GNet Host Functions
// -------------------

// Initializes the GNet
__global__ void initialize(GNet* gnet) {
  gnet->node_put[GID()] = 0;
  gnet->vars_put[GID()] = 0;
  gnet->rbag_pos[GID()] = 0;
  for (u32 i = 0; i < RLEN; ++i) {
    gnet->rbag_buf_A[G_RBAG_LEN / TPG * GID()] = 0;
  }
  for (u32 i = 0; i < RLEN; ++i) {
    gnet->rbag_buf_B[G_RBAG_LEN / TPG * GID()] = 0;
  }
}

GNet* gnet_create() {
  GNet *gnet;
  hipMalloc((void**)&gnet, sizeof(GNet));
  initialize<<<BPG, TPB>>>(gnet);
  //hipMemset(gnet, 0, sizeof(GNet));
  return gnet;
}

u32 gnet_get_rlen(GNet* gnet, u32 turn) {
  u32 rbag_use;
  if (turn % 2 == 0) {
    hipMemcpy(&rbag_use, &gnet->rbag_use_B, sizeof(u32), hipMemcpyDeviceToHost);
  } else {
    hipMemcpy(&rbag_use, &gnet->rbag_use_A, sizeof(u32), hipMemcpyDeviceToHost);
  }
  return rbag_use;
}

u64 gnet_get_itrs(GNet* gnet) {
  u64 itrs;
  hipMemcpy(&itrs, &gnet->itrs, sizeof(u64), hipMemcpyDeviceToHost);
  return itrs;
}

u64 gnet_get_leak(GNet* gnet) {
  u64 leak;
  hipMemcpy(&leak, &gnet->leak, sizeof(u64), hipMemcpyDeviceToHost);
  return leak;
}

void gnet_boot_redex(GNet* gnet, Pair redex) {
  boot_redex<<<BPG, TPB>>>(gnet, redex);
}

void gnet_normalize(GNet* gnet) {
  // Invokes the Evaluator Kernel repeatedly
  u32 turn;
  u64 itrs = 0;
  u32 rlen = 0;
  // NORM
  for (turn = 0; turn < 0xFFFFFFFF; ++turn) {
    //printf("\e[1;1H\e[2J");
    //printf("==================================================== ");
    //printf("TURN: %04x | RLEN: %04x | ITRS: %012llu\n", turn, rlen, itrs);
    //hipDeviceSynchronize();

    evaluator<<<BPG, TPB, sizeof(LNet)>>>(gnet);
    inbetween<<<1, 1>>>(gnet);
    //hipDeviceSynchronize();

    //count_memory<<<BPG, TPB>>>(gnet);
    //hipDeviceSynchronize();

    //print_heatmap<<<1,1>>>(gnet, turn+1);
    //hipDeviceSynchronize();

    itrs = gnet_get_itrs(gnet);
    rlen = gnet_get_rlen(gnet, turn);
    if (rlen == 0) {
      //printf("Completed after %d kernel launches!\n", turn);
      break;
    }
  }
}

// Reads a device node to host
Pair gnet_node_load(GNet* gnet, u32 loc) {
  Pair pair;
  hipMemcpy(&pair, &gnet->node_buf[loc], sizeof(Pair), hipMemcpyDeviceToHost);
  return pair;
}

// Reads a device var to host
Port gnet_vars_load(GNet* gnet, u32 loc) {
  Pair port;
  hipMemcpy(&port, &gnet->vars_buf[loc], sizeof(Port), hipMemcpyDeviceToHost);
  return port;
}

// Like the enter() function, but from host and read-only
Port gnet_peek(GNet* gnet, Port port) {
  while (get_tag(port) == VAR) {
    Port val = gnet_vars_load(gnet, get_val(port));
    if (val == NONE) break;
    port = val;
  }
  return port;
}

// Expands a REF Port.
Port gnet_expand(GNet* gnet, Port port) {
  Port got = gnet_peek(gnet, port);
  //printf("expand %s\n", show_port(got).x);
  while (get_tag(got) == REF) {
    gnet_boot_redex(gnet, new_pair(new_port(REF,get_val(got)), ROOT));
    gnet_normalize(gnet);
    got = gnet_peek(gnet, gnet_vars_load(gnet, get_val(ROOT)));
  }
  return got;
}

// Allocs and creates a node, returning its port.
Port gnet_make_node(GNet* gnet, Tag tag, Port fst, Port snd) {
  Port ret;
  Port* d_ret;
  hipMalloc(&d_ret, sizeof(Port));
  make_node<<<1,1>>>(gnet, tag, fst, snd, d_ret);
  hipMemcpy(&ret, d_ret, sizeof(Port), hipMemcpyDeviceToHost);
  hipFree(d_ret);
  return ret;
}

// Book Loader
// -----------

void book_load(Book* book, u32* buf) {
  // Reads defs_len
  book->defs_len = *buf++;

  //printf("len %d\n", book->defs_len);

  // Parses each def
  for (u32 i = 0; i < book->defs_len; ++i) {
    // Reads fid
    u32 fid = *buf++;

    // Gets def
    Def* def = &book->defs_buf[fid];

    // Reads name
    memcpy(def->name, buf, 256);
    buf += 64;

    // Reads safe flag
    def->safe = *buf++;

    // Reads lengths
    def->rbag_len = *buf++;
    def->node_len = *buf++;
    def->vars_len = *buf++;

    // Reads root
    def->root = *buf++;

    // Reads rbag_buf
    memcpy(def->rbag_buf, buf, 8*def->rbag_len);
    buf += def->rbag_len * 2;

    // Reads node_buf
    memcpy(def->node_buf, buf, 8*def->node_len);
    buf += def->node_len * 2;
  }
}

// Debug Printing
// --------------

__device__ __host__ void put_u32(char* B, u32 val) {
  for (int i = 0; i < 8; i++, val >>= 4) {
    B[8-i-1] = "0123456789ABCDEF"[val & 0xF];
  }
}

__device__ __host__ Show show_port(Port port) {
  // NOTE: this is done like that because sprintf seems not to be working
  Show s;
  switch (get_tag(port)) {
    case VAR: memcpy(s.x, "VAR:", 4); put_u32(s.x+4, get_val(port)); break;
    case REF: memcpy(s.x, "REF:", 4); put_u32(s.x+4, get_val(port)); break;
    case ERA: memcpy(s.x, "ERA:________", 12); break;
    case NUM: memcpy(s.x, "NUM:", 4); put_u32(s.x+4, get_val(port)); break;
    case CON: memcpy(s.x, "CON:", 4); put_u32(s.x+4, get_val(port)); break;
    case DUP: memcpy(s.x, "DUP:", 4); put_u32(s.x+4, get_val(port)); break;
    case OPR: memcpy(s.x, "OPR:", 4); put_u32(s.x+4, get_val(port)); break;
    case SWI: memcpy(s.x, "SWI:", 4); put_u32(s.x+4, get_val(port)); break;
  }
  s.x[12] = '\0';
  return s;
}

__device__ Show show_rule(Rule rule) {
  Show s;
  switch (rule) {
    case LINK: memcpy(s.x, "LINK", 4); break;
    case VOID: memcpy(s.x, "VOID", 4); break;
    case ERAS: memcpy(s.x, "ERAS", 4); break;
    case ANNI: memcpy(s.x, "ANNI", 4); break;
    case COMM: memcpy(s.x, "COMM", 4); break;
    case OPER: memcpy(s.x, "OPER", 4); break;
    case SWIT: memcpy(s.x, "SWIT", 4); break;
    case CALL: memcpy(s.x, "CALL", 4); break;
    default  : memcpy(s.x, "????", 4); break;
  }
  s.x[4] = '\0';
  return s;
}

__device__ void print_rbag(Net* net, TM* tm) {
  printf("RBAG | FST-TREE     | SND-TREE    \n");
  printf("---- | ------------ | ------------\n");
  for (u32 i = 0; i < tm->rbag.hi_end; ++i) {
    Pair redex = tm->rbag.hi_buf[i];
    Pair node1 = get_tag(get_snd(redex)) == CON ? node_load(net, get_val(get_fst(redex))) : 0;
    Pair node2 = get_tag(get_snd(redex)) == CON ? node_load(net, get_val(get_snd(redex))) : 0;
    printf("%04X | %s | %s | hi | (%s %s) ~ (%s %s)\n", i,
      show_port(get_fst(redex)).x,
      show_port(get_snd(redex)).x,
      show_port(peek(net, tm, get_fst(node1))).x,
      show_port(peek(net, tm, get_snd(node1))).x,
      show_port(peek(net, tm, get_fst(node2))).x,
      show_port(peek(net, tm, get_snd(node2))).x);
  }
  for (u32 i = 0; i < tm->rbag.lo_end; ++i) {
    Pair redex = tm->rbag.lo_buf[i%RLEN];
    Pair node1 = get_tag(get_snd(redex)) == CON ? node_load(net, get_val(get_fst(redex))) : 0;
    Pair node2 = get_tag(get_snd(redex)) == CON ? node_load(net, get_val(get_snd(redex))) : 0;
    printf("%04X | %s | %s | hi | (%s %s) ~ (%s %s)\n", i,
      show_port(get_fst(redex)).x,
      show_port(get_snd(redex)).x,
      show_port(peek(net, tm, get_fst(node1))).x,
      show_port(peek(net, tm, get_snd(node1))).x,
      show_port(peek(net, tm, get_fst(node2))).x,
      show_port(peek(net, tm, get_snd(node2))).x);
  }
  printf("==== | ============ | ============\n");
}

__device__ __host__ void print_net(Net* net, u32 ini, u32 end) {
  printf("NODE | PORT-1       | PORT-2      \n");
  printf("---- | ------------ | ------------\n");
  for (u32 i = ini; i < end; ++i) {
    Pair node = node_load(net, i);
    if (node != 0) {
      printf("%04X | %s | %s\n", i, show_port(get_fst(node)).x, show_port(get_snd(node)).x);
    }
  }
  printf("==== | ============ |\n");
  printf("VARS | VALUE        |\n");
  printf("---- | ------------ |\n");
  for (u32 i = ini; i < end; ++i) {
    Port var = vars_load(net,i);
    if (var != 0) {
      printf("%04X | %s |\n", i, show_port(vars_load(net,i)).x);
    }
  }
  printf("==== | ============ |\n");
}

__device__ void pretty_print_numb(Numb word) {
  switch (get_typ(word)) {
    case TY_SYM: {
      switch (get_sym(word)) {
        // types
        case TY_U24: printf("[u24]"); break;
        case TY_I24: printf("[i24]"); break;
        case TY_F24: printf("[f24]"); break;
        // operations
        case OP_ADD: printf("[+]"); break;
        case OP_SUB: printf("[-]"); break;
        case FP_SUB: printf("[:-]"); break;
        case OP_MUL: printf("[*]"); break;
        case OP_DIV: printf("[/]"); break;
        case FP_DIV: printf("[:/]"); break;
        case OP_REM: printf("[%%]"); break;
        case FP_REM: printf("[:%%]"); break;
        case OP_EQ:  printf("[=]"); break;
        case OP_NEQ: printf("[!]"); break;
        case OP_LT:  printf("[<]"); break;
        case OP_GT:  printf("[>]"); break;
        case OP_AND: printf("[&]"); break;
        case OP_OR:  printf("[|]"); break;
        case OP_XOR: printf("[^]"); break;
        case OP_SHL: printf("[<<]"); break;
        case FP_SHL: printf("[:<<]"); break;
        case OP_SHR: printf("[>>]"); break;
        case FP_SHR: printf("[:>>]"); break;
        default:     printf("[?]"); break;
      }
      break;
    }
    case TY_U24: {
      printf("%u", get_u24(word));
      break;
    }
    case TY_I24: {
      printf("%+d", get_i24(word));
      break;
    }
    case TY_F24: {
      if (isinf(get_f24(word))) {
        if (signbit(get_f24(word))) {
          printf("-inf");
        } else {
          printf("+inf");
        }
      } else if (isnan(get_f24(word))) {
        printf("+NaN");
      } else {
        printf("%.7e", get_f24(word));
      }
      break;
    }
    default: {
      switch (get_typ(word)) {
        case OP_ADD: printf("[+0x%07X]", get_u24(word)); break;
        case OP_SUB: printf("[-0x%07X]", get_u24(word)); break;
        case FP_SUB: printf("[:-0x%07X]", get_u24(word)); break;
        case OP_MUL: printf("[*0x%07X]", get_u24(word)); break;
        case OP_DIV: printf("[/0x%07X]", get_u24(word)); break;
        case FP_DIV: printf("[:/0x%07X]", get_u24(word)); break;
        case OP_REM: printf("[%%0x%07X]", get_u24(word)); break;
        case FP_REM: printf("[:%%0x%07X]", get_u24(word)); break;
        case OP_EQ:  printf("[=0x%07X]", get_u24(word)); break;
        case OP_NEQ: printf("[!0x%07X]", get_u24(word)); break;
        case OP_LT:  printf("[<0x%07X]", get_u24(word)); break;
        case OP_GT:  printf("[>0x%07X]", get_u24(word)); break;
        case OP_AND: printf("[&0x%07X]", get_u24(word)); break;
        case OP_OR:  printf("[|0x%07X]", get_u24(word)); break;
        case OP_XOR: printf("[^0x%07X]", get_u24(word)); break;
        case OP_SHL: printf("[<<0x%07X]", get_u24(word)); break;
        case FP_SHL: printf("[:<<0x%07X]", get_u24(word)); break;
        case OP_SHR: printf("[>>0x%07X]", get_u24(word)); break;
        case FP_SHR: printf("[:>>0x%07X]", get_u24(word)); break;
        default:     printf("[?0x%07X]", get_u24(word)); break;
      }
      break;
    }
  }
}

__device__ void pretty_print_port(Net* net, Port port) {
  Port stack[4096];
  stack[0] = port;
  u32 len = 1;
  while (len > 0) {
    if (len > 256) {
      printf("ERROR: result too deep to print. This will be fixed soon(TM)");
      --len;
      continue;
    }
    Port cur = stack[--len];
    switch (get_tag(cur)) {
      case CON: {
        Pair node = node_load(net,get_val(cur));
        Port p2   = get_snd(node);
        Port p1   = get_fst(node);
        printf("(");
        stack[len++] = new_port(ERA, (u32)(')'));
        stack[len++] = p2;
        stack[len++] = new_port(ERA, (u32)(' '));
        stack[len++] = p1;
        break;
      }
      case ERA: {
        if (get_val(cur) != 0) {
          printf("%c", (char)get_val(cur));
        } else {
          printf("*");
        }
        break;
      }
      case VAR: {
        Port got = vars_load(net, get_val(cur));
        if (got != NONE) {
          stack[len++] = got;
        } else {
          printf("x%x", get_val(cur));
        }
        break;
      }
      case NUM: {
        pretty_print_numb(get_val(cur));
        break;
      }
      case DUP: {
        Pair node = node_load(net,get_val(cur));
        Port p2   = get_snd(node);
        Port p1   = get_fst(node);
        printf("{");
        stack[len++] = new_port(ERA, (u32)('}'));
        stack[len++] = p2;
        stack[len++] = new_port(ERA, (u32)(' '));
        stack[len++] = p1;
        break;
      }
      case OPR: {
        Pair node = node_load(net,get_val(cur));
        Port p2   = get_snd(node);
        Port p1   = get_fst(node);
        printf("$(");
        stack[len++] = new_port(ERA, (u32)(')'));
        stack[len++] = p2;
        stack[len++] = new_port(ERA, (u32)(' '));
        stack[len++] = p1;
        break;
      }
      case SWI: {
        Pair node = node_load(net,get_val(cur));
        Port p2   = get_snd(node);
        Port p1   = get_fst(node);
        printf("?(");
        stack[len++] = new_port(ERA, (u32)(')'));
        stack[len++] = p2;
        stack[len++] = new_port(ERA, (u32)(' '));
        stack[len++] = p1;
        break;
      }
      case REF: {
        u32  fid = get_val(cur) & 0xFFFFFFF;
        Def* def = &BOOK.defs_buf[fid];
        printf("@%s", def->name);
        break;
      }
    }
  }
}

__device__ void pretty_print_rbag(Net* net, RBag* rbag) {
  for (u32 i = 0; i < rbag->lo_end; ++i) {
    Pair redex = rbag->lo_buf[i%RLEN];
    if (redex != 0) {
      pretty_print_port(net, get_fst(redex));
      printf(" ~ ");
      pretty_print_port(net, get_snd(redex));
      printf("\n");
    }
  }
  for (u32 i = 0; i < rbag->hi_end; ++i) {
    Pair redex = rbag->hi_buf[i];
    if (redex != 0) {
      pretty_print_port(net, get_fst(redex));
      printf(" ~ ");
      pretty_print_port(net, get_snd(redex));
      printf("\n");
    }
  }
}

__device__ u32 NODE_COUNT;
__device__ u32 VARS_COUNT;

__global__ void count_memory(GNet* gnet) {
  u32 node_count = 0;
  u32 vars_count = 0;
  for (u32 i = GID(); i < G_NODE_LEN; i += TPG) {
    if (gnet->node_buf[i] != 0) ++node_count;
    if (gnet->vars_buf[i] != 0) ++vars_count;
  }

  __shared__ u32 block_node_count;
  __shared__ u32 block_vars_count;

  if (TID() == 0) block_node_count = 0;
  if (TID() == 0) block_vars_count = 0;
  __syncthreads();

  atomicAdd(&block_node_count, node_count);
  atomicAdd(&block_vars_count, vars_count);
  __syncthreads();

  if (TID() == 0) atomicAdd(&NODE_COUNT, block_node_count);
  if (TID() == 0) atomicAdd(&VARS_COUNT, block_vars_count);
}

__global__ void print_heatmap(GNet* gnet, u32 turn) {
  if (GID() > 0) return;

  const char* heatChars[] = {
    //" ", ".", ":", ":",
    //"∴", "⁘", "⁙", "░",
    //"░", "░", "▒", "▒",
    //"▒", "▓", "▓", "▓"
    " ", "1", "2", "3",
    "4", "5", "6", "7",
    "8", "9", "A", "B",
    "C", "D", "E", "F",
  };

  for (u32 bid = 0; bid < BPG; bid++) {
    printf("|");
    for (u32 tid = 0; tid < TPB; tid++) {
      u32 gid = bid * TPB + tid;
      u32 len = 0;
      for (u32 i = 0; i < RLEN; i++) {
        if ( turn % 2 == 0 && gnet->rbag_buf_A[gid * RLEN + i] != 0
          || turn % 2 == 1 && gnet->rbag_buf_B[gid * RLEN + i] != 0) {
          len++;
        }
      }
      u32 pos = gnet->rbag_pos[gid];
      u32 heat = min(len, 0xF);
      printf("%s", heatChars[heat]);
    }
    printf("|\n");
  }
}

__global__ void print_result(GNet* gnet) {
  Net net = vnet_new(gnet, NULL, gnet->turn);
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    printf("Result: ");
    pretty_print_port(&net, enter(&net, NULL, ROOT));
    printf("\n");
  }
}

__global__ void compact(GNet* gnet, Pair* node_out, Port* vars_out) {
  // Ideia: percorrer os nós de forma semelhante a `pretty_print_port`,
  // colocando eles nos buffers em questão usando uma stack.

  // ???
  // if (threadIdx.x == 0 && blockIdx.x == 0) {
  //   Port r = vars_take(gnet, ROOT);
  // }
}

// Demos
// -----

  // stress_test 2^10 x 65536
  //static const u8 BOOK_BUF[] = {6, 0, 0, 0, 0, 0, 0, 0, 109, 97, 105, 110, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 9, 0, 0, 0, 4, 0, 0, 0, 11, 10, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 102, 117, 110, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 1, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 0, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 17, 0, 0, 0, 25, 0, 0, 0, 2, 0, 0, 0, 102, 117, 110, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 33, 0, 0, 0, 4, 0, 0, 0, 11, 0, 0, 1, 0, 0, 0, 0, 3, 0, 0, 0, 102, 117, 110, 95, 95, 67, 49, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 6, 0, 0, 0, 4, 0, 0, 0, 4, 0, 0, 0, 9, 0, 0, 128, 20, 0, 0, 0, 9, 0, 0, 128, 44, 0, 0, 0, 13, 0, 0, 0, 16, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 0, 0, 0, 0, 30, 0, 0, 0, 3, 4, 0, 0, 38, 0, 0, 0, 24, 0, 0, 0, 16, 0, 0, 0, 8, 0, 0, 0, 24, 0, 0, 0, 4, 0, 0, 0, 108, 111, 111, 112, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 1, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 0, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 11, 0, 0, 0, 41, 0, 0, 0, 5, 0, 0, 0, 108, 111, 111, 112, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 33, 0, 0, 0, 0, 0, 0, 0};

  // stress_test 2^18 x 65536
  //static const u8 BOOK_BUF[] = {6, 0, 0, 0, 0, 0, 0, 0, 109, 97, 105, 110, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 9, 0, 0, 0, 4, 0, 0, 0, 11, 18, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 102, 117, 110, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 1, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 0, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 17, 0, 0, 0, 25, 0, 0, 0, 2, 0, 0, 0, 102, 117, 110, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 33, 0, 0, 0, 4, 0, 0, 0, 11, 0, 0, 1, 0, 0, 0, 0, 3, 0, 0, 0, 102, 117, 110, 95, 95, 67, 49, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 6, 0, 0, 0, 4, 0, 0, 0, 4, 0, 0, 0, 9, 0, 0, 128, 20, 0, 0, 0, 9, 0, 0, 128, 44, 0, 0, 0, 13, 0, 0, 0, 16, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 0, 0, 0, 0, 30, 0, 0, 0, 3, 4, 0, 0, 38, 0, 0, 0, 24, 0, 0, 0, 16, 0, 0, 0, 8, 0, 0, 0, 24, 0, 0, 0, 4, 0, 0, 0, 108, 111, 111, 112, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 1, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 0, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 11, 0, 0, 0, 41, 0, 0, 0, 5, 0, 0, 0, 108, 111, 111, 112, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 33, 0, 0, 0, 0, 0, 0, 0};

  // bitonic_sort 2^20
  //static const u8 BOOK_BUF[] = {19, 0, 0, 0, 0, 0, 0, 0, 109, 97, 105, 110, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 2, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 89, 0, 0, 0, 4, 0, 0, 0, 11, 18, 0, 0, 12, 0, 0, 0, 65, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 100, 111, 119, 110, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 9, 0, 0, 0, 4, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 60, 0, 0, 0, 20, 0, 0, 0, 44, 0, 0, 0, 28, 0, 0, 0, 17, 0, 0, 0, 0, 0, 0, 0, 36, 0, 0, 0, 2, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 52, 0, 0, 0, 16, 0, 0, 0, 24, 0, 0, 0, 16, 0, 0, 0, 68, 0, 0, 0, 8, 0, 0, 0, 24, 0, 0, 0, 2, 0, 0, 0, 100, 111, 119, 110, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 13, 0, 0, 0, 8, 0, 0, 0, 4, 0, 0, 0, 25, 0, 0, 128, 60, 0, 0, 0, 25, 0, 0, 128, 84, 0, 0, 0, 13, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 28, 0, 0, 0, 36, 0, 0, 0, 16, 0, 0, 0, 24, 0, 0, 0, 45, 0, 0, 0, 52, 0, 0, 0, 32, 0, 0, 0, 40, 0, 0, 0, 48, 0, 0, 0, 56, 0, 0, 0, 0, 0, 0, 0, 68, 0, 0, 0, 32, 0, 0, 0, 76, 0, 0, 0, 16, 0, 0, 0, 48, 0, 0, 0, 8, 0, 0, 0, 92, 0, 0, 0, 40, 0, 0, 0, 100, 0, 0, 0, 24, 0, 0, 0, 56, 0, 0, 0, 3, 0, 0, 0, 102, 108, 111, 119, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 9, 0, 0, 0, 4, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 60, 0, 0, 0, 20, 0, 0, 0, 44, 0, 0, 0, 28, 0, 0, 0, 33, 0, 0, 0, 0, 0, 0, 0, 36, 0, 0, 0, 2, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 52, 0, 0, 0, 16, 0, 0, 0, 24, 0, 0, 0, 16, 0, 0, 0, 68, 0, 0, 0, 8, 0, 0, 0, 24, 0, 0, 0, 4, 0, 0, 0, 102, 108, 111, 119, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 14, 0, 0, 0, 8, 0, 0, 0, 4, 0, 0, 0, 9, 0, 0, 0, 60, 0, 0, 0, 129, 0, 0, 0, 84, 0, 0, 0, 13, 0, 0, 0, 28, 0, 0, 0, 22, 0, 0, 0, 8, 0, 0, 0, 35, 1, 0, 0, 0, 0, 0, 0, 36, 0, 0, 0, 44, 0, 0, 0, 16, 0, 0, 0, 24, 0, 0, 0, 53, 0, 0, 0, 48, 0, 0, 0, 32, 0, 0, 0, 40, 0, 0, 0, 0, 0, 0, 0, 68, 0, 0, 0, 32, 0, 0, 0, 76, 0, 0, 0, 56, 0, 0, 0, 48, 0, 0, 0, 8, 0, 0, 0, 92, 0, 0, 0, 40, 0, 0, 0, 100, 0, 0, 0, 16, 0, 0, 0, 108, 0, 0, 0, 24, 0, 0, 0, 56, 0, 0, 0, 5, 0, 0, 0, 103, 101, 110, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 4, 0, 0, 0, 2, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 8, 0, 0, 0, 20, 0, 0, 0, 8, 0, 0, 0, 28, 0, 0, 0, 49, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 6, 0, 0, 0, 103, 101, 110, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 12, 0, 0, 0, 6, 0, 0, 0, 4, 0, 0, 0, 41, 0, 0, 128, 68, 0, 0, 0, 41, 0, 0, 128, 84, 0, 0, 0, 13, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 29, 0, 0, 0, 60, 0, 0, 0, 38, 0, 0, 0, 54, 0, 0, 0, 59, 2, 0, 0, 46, 0, 0, 0, 35, 1, 0, 0, 16, 0, 0, 0, 59, 2, 0, 0, 24, 0, 0, 0, 32, 0, 0, 0, 40, 0, 0, 0, 0, 0, 0, 0, 76, 0, 0, 0, 16, 0, 0, 0, 32, 0, 0, 0, 8, 0, 0, 0, 92, 0, 0, 0, 24, 0, 0, 0, 40, 0, 0, 0, 7, 0, 0, 0, 109, 97, 105, 110, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 2, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 41, 0, 0, 0, 4, 0, 0, 0, 11, 18, 0, 0, 12, 0, 0, 0, 11, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 109, 97, 105, 110, 95, 95, 67, 49, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 3, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 73, 0, 0, 0, 4, 0, 0, 0, 11, 18, 0, 0, 12, 0, 0, 0, 11, 0, 0, 0, 20, 0, 0, 0, 57, 0, 0, 0, 0, 0, 0, 0, 9, 0, 0, 0, 115, 111, 114, 116, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 9, 0, 0, 0, 4, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 60, 0, 0, 0, 20, 0, 0, 0, 44, 0, 0, 0, 28, 0, 0, 0, 81, 0, 0, 0, 0, 0, 0, 0, 36, 0, 0, 0, 2, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 52, 0, 0, 0, 16, 0, 0, 0, 24, 0, 0, 0, 16, 0, 0, 0, 68, 0, 0, 0, 8, 0, 0, 0, 24, 0, 0, 0, 10, 0, 0, 0, 115, 111, 114, 116, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 17, 0, 0, 0, 9, 0, 0, 0, 4, 0, 0, 0, 25, 0, 0, 0, 60, 0, 0, 0, 73, 0, 0, 128, 92, 0, 0, 0, 73, 0, 0, 128, 116, 0, 0, 0, 13, 0, 0, 0, 36, 0, 0, 0, 22, 0, 0, 0, 29, 0, 0, 0, 35, 1, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 16, 0, 0, 0, 44, 0, 0, 0, 52, 0, 0, 0, 24, 0, 0, 0, 32, 0, 0, 0, 40, 0, 0, 0, 48, 0, 0, 0, 0, 0, 0, 0, 68, 0, 0, 0, 40, 0, 0, 0, 76, 0, 0, 0, 84, 0, 0, 0, 48, 0, 0, 0, 56, 0, 0, 0, 64, 0, 0, 0, 8, 0, 0, 0, 100, 0, 0, 0, 11, 0, 0, 0, 108, 0, 0, 0, 24, 0, 0, 0, 56, 0, 0, 0, 16, 0, 0, 0, 124, 0, 0, 0, 11, 1, 0, 0, 132, 0, 0, 0, 32, 0, 0, 0, 64, 0, 0, 0, 11, 0, 0, 0, 115, 117, 109, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 4, 0, 0, 0, 2, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 8, 0, 0, 0, 20, 0, 0, 0, 8, 0, 0, 0, 28, 0, 0, 0, 97, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 12, 0, 0, 0, 115, 117, 109, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 10, 0, 0, 0, 6, 0, 0, 0, 4, 0, 0, 0, 89, 0, 0, 128, 36, 0, 0, 0, 89, 0, 0, 128, 68, 0, 0, 0, 13, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 28, 0, 0, 0, 32, 0, 0, 0, 16, 0, 0, 0, 24, 0, 0, 0, 0, 0, 0, 0, 44, 0, 0, 0, 16, 0, 0, 0, 54, 0, 0, 0, 3, 4, 0, 0, 62, 0, 0, 0, 40, 0, 0, 0, 32, 0, 0, 0, 8, 0, 0, 0, 76, 0, 0, 0, 24, 0, 0, 0, 40, 0, 0, 0, 13, 0, 0, 0, 115, 119, 97, 112, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 7, 0, 0, 0, 3, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 44, 0, 0, 0, 20, 0, 0, 0, 28, 0, 0, 0, 113, 0, 0, 0, 121, 0, 0, 0, 0, 0, 0, 0, 36, 0, 0, 0, 8, 0, 0, 0, 16, 0, 0, 0, 8, 0, 0, 0, 52, 0, 0, 0, 0, 0, 0, 0, 16, 0, 0, 0, 14, 0, 0, 0, 115, 119, 97, 112, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 2, 0, 0, 0, 4, 0, 0, 0, 0, 0, 0, 0, 12, 0, 0, 0, 8, 0, 0, 0, 20, 0, 0, 0, 8, 0, 0, 0, 0, 0, 0, 0, 15, 0, 0, 0, 115, 119, 97, 112, 95, 95, 67, 49, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 4, 0, 0, 0, 2, 0, 0, 0, 4, 0, 0, 0, 2, 0, 0, 0, 12, 0, 0, 0, 0, 0, 0, 0, 20, 0, 0, 0, 8, 0, 0, 0, 28, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 16, 0, 0, 0, 119, 97, 114, 112, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 9, 0, 0, 0, 4, 0, 0, 0, 4, 0, 0, 0, 15, 0, 0, 0, 52, 0, 0, 0, 20, 0, 0, 0, 28, 0, 0, 0, 137, 0, 0, 0, 145, 0, 0, 0, 0, 0, 0, 0, 36, 0, 0, 0, 8, 0, 0, 0, 44, 0, 0, 0, 16, 0, 0, 0, 24, 0, 0, 0, 16, 0, 0, 0, 60, 0, 0, 0, 8, 0, 0, 0, 68, 0, 0, 0, 0, 0, 0, 0, 24, 0, 0, 0, 17, 0, 0, 0, 119, 97, 114, 112, 95, 95, 67, 48, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 12, 0, 0, 0, 6, 0, 0, 0, 4, 0, 0, 0, 105, 0, 0, 0, 76, 0, 0, 0, 13, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 29, 0, 0, 0, 52, 0, 0, 0, 38, 0, 0, 0, 24, 0, 0, 0, 3, 15, 0, 0, 46, 0, 0, 0, 0, 0, 0, 0, 16, 0, 0, 0, 62, 0, 0, 0, 40, 0, 0, 0, 3, 18, 0, 0, 70, 0, 0, 0, 16, 0, 0, 0, 32, 0, 0, 0, 32, 0, 0, 0, 84, 0, 0, 0, 24, 0, 0, 0, 92, 0, 0, 0, 8, 0, 0, 0, 40, 0, 0, 0, 18, 0, 0, 0, 119, 97, 114, 112, 95, 95, 67, 49, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 21, 0, 0, 0, 12, 0, 0, 0, 4, 0, 0, 0, 129, 0, 0, 128, 92, 0, 0, 0, 129, 0, 0, 128, 132, 0, 0, 0, 13, 0, 0, 0, 20, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 28, 0, 0, 0, 36, 0, 0, 0, 16, 0, 0, 0, 24, 0, 0, 0, 44, 0, 0, 0, 52, 0, 0, 0, 32, 0, 0, 0, 40, 0, 0, 0, 61, 0, 0, 0, 68, 0, 0, 0, 48, 0, 0, 0, 56, 0, 0, 0, 76, 0, 0, 0, 84, 0, 0, 0, 64, 0, 0, 0, 72, 0, 0, 0, 80, 0, 0, 0, 88, 0, 0, 0, 8, 0, 0, 0, 100, 0, 0, 0, 56, 0, 0, 0, 108, 0, 0, 0, 40, 0, 0, 0, 116, 0, 0, 0, 24, 0, 0, 0, 124, 0, 0, 0, 72, 0, 0, 0, 88, 0, 0, 0, 0, 0, 0, 0, 140, 0, 0, 0, 48, 0, 0, 0, 148, 0, 0, 0, 32, 0, 0, 0, 156, 0, 0, 0, 16, 0, 0, 0, 164, 0, 0, 0, 64, 0, 0, 0, 80, 0, 0, 0};

//COMPILED_BOOK_BUF//

// Main
// ----

#ifdef IO
void do_run_io(GNet* gnet, Book* book, Port port);
#endif

extern "C" void hvm_cu(u32* book_buffer, bool return_output) {
  // Start the timer
  clock_t start = clock();

  // Loads the Book
  Book* book = (Book*)malloc(sizeof(Book));
  if (book_buffer) {
    book_load(book, (u32*)book_buffer);
    hipMemcpyToSymbol(HIP_SYMBOL(BOOK), book, sizeof(Book));
  }

  // Configures Shared Memory Size
  hipFuncSetAttribute(reinterpret_cast<const void*>(evaluator), hipFuncAttributeMaxDynamicSharedMemorySize, sizeof(LNet));

  // Creates a new GNet
  GNet* gnet = gnet_create();

  // Boots root redex, to expand @main
  gnet_boot_redex(gnet, new_pair(new_port(REF, 0), ROOT));

  #ifdef IO
  do_run_io(gnet, book, ROOT);
  #else
  gnet_normalize(gnet);
  #endif

  hipDeviceSynchronize();

  // Stops the timer
  clock_t end = clock();
  double duration = ((double)(end - start)) / CLOCKS_PER_SEC;

  // Prints the result
  // If `output` is set, the Rust implementation will print the net
  if (!return_output) {
    print_result<<<1,1>>>(gnet);
  }

  // Reports errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch kernels. Error code: %s.\n", hipGetErrorString(err));
    if (err == hipErrorInvalidConfiguration) {
      fprintf(stderr, "Note: for now, HVM-CUDA requires a GPU with at least 128 KB of L1 cache per SM.\n");
    }
    exit(EXIT_FAILURE);
  }

  // If `output` is set, copy the memory from the net into the Rust implementation
  if (return_output) {
    // hipMemcpy(output, gnet, sizeof(GNet), hipMemcpyDeviceToHost);
  }

  // Prints entire memdump
  //{
    //// Allocate host memory for the net
    //GNet *h_gnet = (GNet*)malloc(sizeof(GNet));

    //// Copy the net from device to host
    //hipMemcpy(h_gnet, gnet, sizeof(GNet), hipMemcpyDeviceToHost);

    //// Create a Net view of the host GNet
    //Net net;
    //net.g_node_buf = h_gnet->node_buf;
    //net.g_vars_buf = h_gnet->vars_buf;

    //// Print the net
    //print_net(&net, L_NODE_LEN, G_NODE_LEN);

    //// Free host memory
    //free(h_gnet);
  //}

  // Gets interaction count
  //hipMemcpy(&itrs, &gnet->itrs, sizeof(u64), hipMemcpyDeviceToHost);

  // Prints interactions, time and MIPS
  // If `output` is set, the Rust implementation will print the net
  if (!return_output) {
    printf("- ITRS: %llu\n", gnet_get_itrs(gnet));
    printf("- LEAK: %llu\n", gnet_get_leak(gnet));
    printf("- TIME: %.2fs\n", duration);
    printf("- MIPS: %.2f\n", (double)gnet_get_itrs(gnet) / duration / 1000000.0);
  }
}

#ifdef WITH_MAIN
int main() {
  hvm_cu((u32*)BOOK_BUF, NULL);
  return 0;
}
#endif
