#include "hip/hip_runtime.h"
#include "hvm.cu"

// Readback: λ-Encoded Ctr
struct Ctr {
  u32  tag;
  u32  args_len;
  Port args_buf[16];
};

// Readback: Tuples
struct Tup {
  u32  elem_len;
  Port elem_buf[8];
};

// Readback: λ-Encoded Str (UTF-32)
// FIXME: this is actually ASCII :|
// FIXME: remove len limit
struct Str {
  u32  text_len;
  char text_buf[256];
};

// Readback: λ-Encoded list of bytes
typedef struct Bytes {
  u32  len;
  char *buf;
} Bytes;

#define MAX_BYTES 256

// IO Magic Number
#define IO_MAGIC_0 0xD0CA11
#define IO_MAGIC_1 0xFF1FF1

// IO Tags
#define IO_DONE 0
#define IO_CALL 1

// List Type
#define LIST_NIL  0
#define LIST_CONS 1

// Readback
// --------

// Reads back a λ-Encoded constructor from device to host.
// Encoding: λt ((((t TAG) arg0) arg1) ...)
Ctr gnet_readback_ctr(GNet* gnet, Port port) {
  Ctr ctr;
  ctr.tag = -1;
  ctr.args_len = 0;

  // Loads root lambda
  Port lam_port = gnet_expand(gnet, port);
  if (get_tag(lam_port) != CON) return ctr;
  Pair lam_node = gnet_node_load(gnet, get_val(lam_port));

  // Loads first application
  Port app_port = gnet_expand(gnet, get_fst(lam_node));
  if (get_tag(app_port) != CON) return ctr;
  Pair app_node = gnet_node_load(gnet, get_val(app_port));

  // Loads first argument (as the tag)
  Port arg_port = gnet_expand(gnet, get_fst(app_node));
  if (get_tag(arg_port) != NUM) return ctr;
  ctr.tag = get_u24(get_val(arg_port));

  // Loads remaining arguments
  while (TRUE) {
    app_port = gnet_expand(gnet, get_snd(app_node));
    if (get_tag(app_port) != CON) break;
    app_node = gnet_node_load(gnet, get_val(app_port));
    arg_port = gnet_expand(gnet, get_fst(app_node));
    ctr.args_buf[ctr.args_len++] = arg_port;
  }

  return ctr;
}

// Reads back a tuple of at most `size` elements. Tuples are
// (right-nested con nodes) (CON 1 (CON 2 (CON 3 (...))))
// The provided `port` should be `expanded` before calling.
Tup gnet_readback_tup(GNet* gnet, Port port, u32 size) {
  Tup tup;
  tup.elem_len = 0;

  // Loads remaining arguments
  while (get_tag(port) == CON && (tup.elem_len + 1 < size)) {
    Pair node = gnet_node_load(gnet, get_val(port));
    tup.elem_buf[tup.elem_len++] = gnet_expand(gnet, get_fst(node));

    port = gnet_expand(gnet, get_snd(node));
  }

  tup.elem_buf[tup.elem_len++] = port;

  return tup;
}


// Reads back a UTF-32 (truncated to 24 bits) string.
// Since unicode scalars can fit in 21 bits, HVM's u24
// integers can contain any unicode scalar value.
// Encoding:
// - λt (t NIL)
// - λt (((t CONS) head) tail)
Str gnet_readback_str(GNet* gnet, Port port) {
  // Result
  Str str;
  str.text_len = 0;

  // Readback loop
  while (TRUE) {
    // Normalizes the net
    gnet_normalize(gnet);

    // Reads the λ-Encoded Ctr
    Ctr ctr = gnet_readback_ctr(gnet, gnet_peek(gnet, port));

    // Reads string layer
    switch (ctr.tag) {
      case LIST_NIL: {
        break;
      }
      case LIST_CONS: {
        if (ctr.args_len != 2) break;
        if (get_tag(ctr.args_buf[0]) != NUM) break;
        if (str.text_len >= 255) { printf("ERROR: for now, HVM can only readback strings of length <256."); break; }

        str.text_buf[str.text_len++] = get_u24(get_val(ctr.args_buf[0]));
        gnet_boot_redex(gnet, new_pair(ctr.args_buf[1], ROOT));
        port = ROOT;
        continue;
      }
    }
    break;
  }

  str.text_buf[str.text_len] = '\0';

  return str;
}

// Converts a Port into a list of bytes.
// Encoding:
// - λt (t NIL)
// - λt (((t CONS) head) tail)
Bytes gnet_readback_bytes(GNet* gnet, Port port) {
  // Result
  Bytes bytes;
  bytes.buf = (char*) malloc(sizeof(char) * MAX_BYTES);
  bytes.len = 0;

  // Readback loop
  while (TRUE) {
    // Normalizes the net
    gnet_normalize(gnet);

    // Reads the λ-Encoded Ctr
    Ctr ctr = gnet_readback_ctr(gnet, gnet_peek(gnet, port));

    // Reads string layer
    switch (ctr.tag) {
      case LIST_NIL: {
        break;
      }
      case LIST_CONS: {
        if (ctr.args_len != 2) break;
        if (get_tag(ctr.args_buf[0]) != NUM) break;
        if (bytes.len >= MAX_BYTES) { printf("ERROR: for now, HVM can only readback list of bytes of length <%u.", MAX_BYTES); break; }

        bytes.buf[bytes.len++] = get_u24(get_val(ctr.args_buf[0]));
        gnet_boot_redex(gnet, new_pair(ctr.args_buf[1], ROOT));
        port = ROOT;
        continue;
      }
    }
    break;
  }

  bytes.buf[bytes.len] = '\0';

  return bytes;
}

/// Returns a λ-Encoded Ctr for a NIL: λt (t NIL)
/// Should only be called within `inject_bytes`, as a previous call
/// to `get_resources` is expected.
__device__ Port inject_nil(Net* net, TM* tm) {
  u32 v1 = tm->vloc[0];

  u32 n1 = tm->nloc[0];
  u32 n2 = tm->nloc[1];

  vars_create(net, v1, NONE);
  Port var = new_port(VAR, v1);

  node_create(net, n1, new_pair(new_port(NUM, new_u24(LIST_NIL)), var));
  node_create(net, n2, new_pair(new_port(CON, n1), var));

  return new_port(CON, n2);
}

/// Returns a λ-Encoded Ctr for a CONS: λt (((t CONS) head) tail)
/// Should only be called within `inject_bytes`, as a previous call
/// to `get_resources` is expected.
/// The `char_idx` parameter is used to offset the vloc and nloc
/// allocations, otherwise they would conflict with each other on
/// subsequent calls.
__device__ Port inject_cons(Net* net, TM* tm, Port head, Port tail, u32 char_idx) {
  u32 v1 = tm->vloc[1 + char_idx];

  u32 n1 = tm->nloc[2 + char_idx * 4 + 0];
  u32 n2 = tm->nloc[2 + char_idx * 4 + 1];
  u32 n3 = tm->nloc[2 + char_idx * 4 + 2];
  u32 n4 = tm->nloc[2 + char_idx * 4 + 3];

  vars_create(net, v1, NONE);
  Port var = new_port(VAR, v1);

  node_create(net, n1, new_pair(tail, var));
  node_create(net, n2, new_pair(head, new_port(CON, n1)));
  node_create(net, n3, new_pair(new_port(NUM, new_u24(LIST_CONS)), new_port(CON, n2)));
  node_create(net, n4, new_pair(new_port(CON, n3), var));

  return new_port(CON, n4);
}

// Converts a list of bytes to a Port.
// Encoding:
// - λt (t NIL)
// - λt (((t CONS) head) tail)
__device__ Port inject_bytes(Net* net, TM* tm, Bytes *bytes) {
  // Allocate all resources up front:
  // - NIL needs  2 nodes & 1 var
  // - CONS needs 4 nodes & 1 var
  u32 len = bytes->len;
  if (!get_resources(net, tm, 0, 2 + 4 * len, 1 + len)) {
    printf("inject_bytes: failed to get resources\n");
    return new_port(ERA, 0);
  }

  Port port = inject_nil(net, tm);

  for (u32 i = 0; i < len; i++) {
    Port byte = new_port(NUM, new_u24(bytes->buf[len - i - 1]));
    port = inject_cons(net, tm, byte, port, i);
  }

  return port;
}

__global__ void make_bytes_port(GNet* gnet, Bytes bytes, Port* ret) {
  if (GID() == 0) {
    TM tm;
    Net net = vnet_new(gnet, NULL, gnet->turn);
    *ret = inject_bytes(&net, &tm, &bytes);
  }
}

// Converts a list of bytes to a Port.
// Encoding:
// - λt (t NIL)
// - λt (((t CONS) head) tail)
Port gnet_inject_bytes(GNet* gnet, Bytes *bytes) {
  Port* d_ret;
  hipMalloc(&d_ret, sizeof(Port));

  Bytes cu_bytes;
  cu_bytes.len = bytes->len;

  hipMalloc(&cu_bytes.buf, sizeof(char) * cu_bytes.len);
  hipMemcpy(cu_bytes.buf, bytes->buf, sizeof(char) * cu_bytes.len, hipMemcpyHostToDevice);

  make_bytes_port<<<1,1>>>(gnet, cu_bytes, d_ret);

  Port ret;
  hipMemcpy(&ret, d_ret, sizeof(Port), hipMemcpyDeviceToHost);
  hipFree(d_ret);
  hipFree(cu_bytes.buf);

  return ret;
}

// Primitive IO Fns
// -----------------

// Open file pointers. Indices into this array
// are used as "file descriptors".
// Indices 0 1 and 2 are reserved.
// - 0 -> stdin
// - 1 -> stdout
// - 2 -> stderr
static FILE* FILE_POINTERS[256];

// Converts a NUM port (file descriptor) to file pointer.
FILE* readback_file(Port port) {
  if (get_tag(port) != NUM) {
    fprintf(stderr, "non-num where file descriptor was expected: %s\n", show_port(port).x);
    return NULL;
  }

  u32 idx = get_u24(get_val(port));

  if (idx == 0) return stdin;
  if (idx == 1) return stdout;
  if (idx == 2) return stderr;

  FILE* fp = FILE_POINTERS[idx];
  if (fp == NULL) {
    fprintf(stderr, "invalid file descriptor\n");
    return NULL;
  }

  return fp;
}

// Reads from a file a specified number of bytes.
// `argm` is a tuple of (file_descriptor, num_bytes).
Port io_read(GNet* gnet, Port argm) {
  Tup tup = gnet_readback_tup(gnet, argm, 2);
  if (tup.elem_len != 2) {
    fprintf(stderr, "io_read: expected 2-tuple\n");
    return new_port(ERA, 0);
  }

  FILE* fp = readback_file(tup.elem_buf[0]);
  u32 num_bytes = get_u24(get_val(tup.elem_buf[1]));

  if (fp == NULL) {
    fprintf(stderr, "io_read: invalid file descriptor\n");
    return new_port(ERA, 0);
  }

  /// Read a string.
  Bytes bytes;
  bytes.buf = (char*) malloc(sizeof(char) * num_bytes);
  bytes.len = fread(bytes.buf, sizeof(char), num_bytes, fp);

  if ((bytes.len != num_bytes) && ferror(fp)) {
    fprintf(stderr, "io_read: failed to read\n");
    free(bytes.buf);
    return new_port(ERA, 0);
  }

  // Convert it to a port.
  Port ret = gnet_inject_bytes(gnet, &bytes);
  free(bytes.buf);
  return ret;
}

// Opens a file with the provided mode.
// `argm` is a tuple (CON node) of the
// file name and mode as strings.
Port io_open(GNet* gnet, Port argm) {
  Tup tup = gnet_readback_tup(gnet, argm, 2);
  if (tup.elem_len != 2) {
    fprintf(stderr, "io_open: expected 2-tuple\n");
    return new_port(ERA, 0);
  }

  Str name = gnet_readback_str(gnet, tup.elem_buf[0]);
  Str mode = gnet_readback_str(gnet, tup.elem_buf[1]);

  for (u32 fd = 3; fd < sizeof(FILE_POINTERS); fd++) {
    if (FILE_POINTERS[fd] == NULL) {
      FILE_POINTERS[fd] = fopen(name.text_buf, mode.text_buf);
      return new_port(NUM, new_u24(fd));
    }
  }

  fprintf(stderr, "io_open: too many open files\n");
  return new_port(ERA, 0);
}

// Closes a file, reclaiming the file descriptor.
Port io_close(GNet* gnet, Port argm) {
  FILE* fp = readback_file(argm);
  if (fp == NULL) {
    fprintf(stderr, "io_close: invalid file descriptor\n");
    return new_port(ERA, 0);
  }

  int err = fclose(fp) != 0;
  if (err != 0) {
    fprintf(stderr, "io_close: failed to close: %i\n", err);
    return new_port(ERA, 0);
  }

  FILE_POINTERS[get_u24(get_val(argm))] = NULL;
  return new_port(ERA, 0);
}

// Flushes an output stream.
Port io_flush(GNet* gnet, Port argm) {
  FILE* fp = readback_file(argm);
  if (fp == NULL) {
    fprintf(stderr, "io_flush: invalid file descriptor\n");
    return new_port(ERA, 0);
  }

  int err = fflush(fp) != 0;
  if (err != 0) {
    fprintf(stderr, "io_flush: failed to flush: %i\n", err);
    return new_port(ERA, 0);
  }

  return new_port(ERA, 0);
}

// Writes a list of bytes to a file.
// `argm` is a tuple (CON node) of the
// file descriptor and list of bytes to write.
Port io_write(GNet* gnet, Port argm) {
  Tup tup = gnet_readback_tup(gnet, argm, 2);
  if (tup.elem_len != 2) {
    fprintf(stderr, "io_write: expected 2-tuple\n");
    return new_port(ERA, 0);
  }

  FILE* fp = readback_file(tup.elem_buf[0]);
  Bytes bytes = gnet_readback_bytes(gnet, tup.elem_buf[1]);

  if (fp == NULL) {
    fprintf(stderr, "io_write: invalid file descriptor\n");
    free(bytes.buf);
    return new_port(ERA, 0);
  }

  if (fwrite(bytes.buf, sizeof(char), bytes.len, fp) != bytes.len) {
    fprintf(stderr, "io_write: failed to write\n");
  }

  free(bytes.buf);
  return new_port(ERA, 0);
}

// Seeks to a position in a file.
// `argm` is a 3-tuple (CON fd (CON offset whence)), where
// - fd is a file descriptor
// - offset is a signed byte offset
// - whence is what that offset is relative to:
//    - 0 (SEEK_SET): beginning of file
//    - 1 (SEEK_CUR): current position of the file pointer
//    - 2 (SEEK_END): end of the file
Port io_seek(GNet* gnet, Port argm) {
  Tup tup = gnet_readback_tup(gnet, argm, 3);
  if (tup.elem_len != 3) {
    fprintf(stderr, "io_seek: expected 3-tuple\n");
    return new_port(ERA, 0);
  }

  FILE* fp = readback_file(tup.elem_buf[0]);
  i32 offset = get_i24(get_val(tup.elem_buf[1]));
  u32 whence = get_i24(get_val(tup.elem_buf[2]));

  if (fp == NULL) {
    fprintf(stderr, "io_write: invalid file descriptor\n");
    return new_port(ERA, 0);
  }

  int cwhence;
  switch (whence) {
    case 0: cwhence = SEEK_SET; break;
    case 1: cwhence = SEEK_CUR; break;
    case 2: cwhence = SEEK_END; break;
    default:
      fprintf(stderr, "io_seek: invalid whence\n");
      return new_port(ERA, 0);
  }

  if (fseek(fp, offset, cwhence) != 0) {
    fprintf(stderr, "io_seek: failed to seek\n");
  }

  return new_port(ERA, 0);
}

// Returns the current time as a tuple of the high
// and low 24 bits of a 48-bit nanosecond timestamp.
Port io_get_time(GNet* gnet, Port argm) {
  // Get the current time in nanoseconds
  u64 time_ns = time64();
  // Encode the time as a 64-bit unsigned integer
  u32 time_hi = (u32)(time_ns >> 24) & 0xFFFFFFF;
  u32 time_lo = (u32)(time_ns & 0xFFFFFFF);
  // Return the encoded time
  return gnet_make_node(gnet, CON, new_port(NUM, new_u24(time_hi)), new_port(NUM, new_u24(time_lo)));
}

// Sleeps.
// `argm` is a tuple (CON node) of the high and low
// 24 bits for a 48-bit duration in nanoseconds.
Port io_sleep(GNet* gnet, Port argm) {
  Tup tup = gnet_readback_tup(gnet, argm, 2);
  if (tup.elem_len != 2) {
    fprintf(stderr, "io_sleep: expected 3-tuple\n");
    return new_port(ERA, 0);
  }

  // Get the sleep duration node
  Pair dur_node = gnet_node_load(gnet, get_val(argm));
  // Get the high and low 24-bit parts of the duration
  u32 dur_hi = get_u24(get_val(tup.elem_buf[0]));
  u32 dur_lo = get_u24(get_val(tup.elem_buf[1]));
  // Combine into a 48-bit duration in nanoseconds
  u64 dur_ns = (((u64)dur_hi) << 24) | dur_lo;
  // Sleep for the specified duration
  struct timespec ts;
  ts.tv_sec = dur_ns / 1000000000;
  ts.tv_nsec = dur_ns % 1000000000;
  nanosleep(&ts, NULL);
  // Return an eraser
  return new_port(ERA, 0);
}

void book_init(Book* book) {
  book->ffns_buf[book->ffns_len++] = (FFn){"READ", io_read};
  book->ffns_buf[book->ffns_len++] = (FFn){"OPEN", io_open};
  book->ffns_buf[book->ffns_len++] = (FFn){"CLOSE", io_close};
  book->ffns_buf[book->ffns_len++] = (FFn){"FLUSH", io_flush};
  book->ffns_buf[book->ffns_len++] = (FFn){"WRITE", io_write};
  book->ffns_buf[book->ffns_len++] = (FFn){"SEEK", io_seek};
  book->ffns_buf[book->ffns_len++] = (FFn){"GET_TIME", io_get_time};
  book->ffns_buf[book->ffns_len++] = (FFn){"SLEEP", io_sleep};

  hipMemcpyToSymbol(HIP_SYMBOL(BOOK), book, sizeof(Book));
}

// Monadic IO Evaluator
// ---------------------

// Runs an IO computation.
void do_run_io(GNet* gnet, Book* book, Port port) {
  book_init(book);

  setlinebuf(stdout);
  setlinebuf(stderr);

  // IO loop
  while (TRUE) {
    // Normalizes the net
    gnet_normalize(gnet);

    // Reads the λ-Encoded Ctr
    Ctr ctr = gnet_readback_ctr(gnet, gnet_peek(gnet, port));

    // Checks if IO Magic Number is a CON
    if (get_tag(ctr.args_buf[0]) != CON) {
      break;
    }

    // Checks the IO Magic Number
    Pair io_magic = gnet_node_load(gnet, get_val(ctr.args_buf[0]));
    //printf("%08x %08x\n", get_u24(get_val(get_fst(io_magic))), get_u24(get_val(get_snd(io_magic))));
    if (get_val(get_fst(io_magic)) != new_u24(IO_MAGIC_0) || get_val(get_snd(io_magic)) != new_u24(IO_MAGIC_1)) {
      break;
    }

    switch (ctr.tag) {
      case IO_CALL: {
        Str  func = gnet_readback_str(gnet, ctr.args_buf[1]);
        FFn* ffn  = NULL;
        // FIXME: optimize this linear search
        for (u32 fid = 0; fid < book->ffns_len; ++fid) {
          if (strcmp(func.text_buf, book->ffns_buf[fid].name) == 0) {
            ffn = &book->ffns_buf[fid];
            break;
          }
        }
        if (ffn == NULL) {
          fprintf(stderr, "Unknown IO func '%s'\n", func.text_buf);
          break;
        }

        Port argm = ctr.args_buf[2];
        Port cont = ctr.args_buf[3];
        Port ret  = ffn->func(gnet, argm);

        Port p = gnet_make_node(gnet, CON, ret, ROOT);
        gnet_boot_redex(gnet, new_pair(p, cont));
        port = ROOT;
        continue;
      }
      case IO_DONE: {
        break;
      }
    }
    break;
  }
}
